#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

static uint * tout_zeroiser = UNIFORME_C(1);

PAS_OPTIMISER()
uint __SI_NAN(Mdl_t * mdl)
{
	float * _gpu = gpu_vers_cpu<float>(mdl->p__d[C-1], mdl->inst_POIDS[C-1]);
	//
	uint vrai = 0;
	FOR(0, i, mdl->inst_POIDS[C-1]) {
		if (_gpu[i] != _gpu[i]) {
			vrai = 1;
			break;
		}
	}
	//
	free(_gpu);
	//
	return vrai;
};

PAS_OPTIMISER()
void __interne_optimiser(
	Mdl_t * mdl,
	uint t0, uint t1,
	float * alpha, float div,
	uint methode, uint I,
	uint ** masque,
	uint ** masque_opti,
	uint PERTURBATIONS,
	uint zero_accumulation_tous_les[C],
	uint _t_MODE, uint GRAINE)
{
	mdl_zero_deriv_gpu(mdl, tout_zeroiser);
	//
	//	Cree les listes pour les `hist` si un opti en a besoin 
	Opti_classe_t opti_classe;
	if      (methode == SGD)     opti_classe.sgd     = (uint)NULL;
	else if (methode == RMSPROP) opti_classe.rmsprop = cree_rmsprop(mdl);
	else if (methode == ADAM)    opti_classe.adam    = cree_adam(mdl);
	else ERR("Pas de methode %i d'optimisation", methode);
	
	//	Plumer grad pour mieux y voire
	mdl_plume_grad(mdl, t0, t1, _t_MODE, GRAINE);
	
	/* ------- Optimisation ----------- */
	uint zeroiser[C];
	FOR(0, i, I) {
		//
		FOR(0, j, C) {
			if (i % zero_accumulation_tous_les[j] == 0)
				zeroiser[j] = 1;
			else
				zeroiser[j] = 0;
		}
		// !!! i=0 ne sera pas optimisé !!!
		//if (i != 0)	//on optimise pas au premiere, pour voire le res avant opti
		//{
		//perturber(mdl, PERTURBATIONS);
		uint EXACTE = 0;
		mdl_aller_retour(mdl, t0, t1, MODE_MAXIMALE, _t_MODE, GRAINE, EXACTE);
		//}

		if (__SI_NAN(mdl)) ERR("Des nan dans mdl");
		
		//	--------- * Optimisation * -------------
#define optimiser_la_couche zeroiser
		if (i != 0) {
			if (methode == SGD)     opti_simple (zero_accumulation_tous_les, optimiser_la_couche, mdl, alpha, div, masque, masque_opti);
			if (methode == RMSPROP) opti_rmsprop(zero_accumulation_tous_les, optimiser_la_couche, mdl, opti_classe.rmsprop, alpha, div, masque, masque_opti);
			if (methode == ADAM)    opti_adam   (zero_accumulation_tous_les, optimiser_la_couche, mdl, opti_classe.adam,    alpha, div, masque, masque_opti);
		}
		//
		mdl_zero_deriv_gpu(mdl, zeroiser);
		//
		mdl_normer_les_filtres(mdl);
		//
		/*	mdl_poids_gpu_vers_cpu(mdl);
		FOR(0, c, C) regulariser_inst[mdl->insts[c]](mdl, c);
			mdl_poids_cpu_vers_gpu(mdl);*/
		//
		if (i % /*100*/50 == 0 || i == 1 || i == I-1) {
			//mdl_plume_grad(mdl, t0, t1, _t_MODE, GRAINE);
			//
			float __pred = mdl_pred (mdl, t0, t1, MODE_MAXIMALE, _t_MODE, GRAINE);
			float _score = mdl_score(mdl, t0, t1, MODE_MAXIMALE, _t_MODE, GRAINE);
			//
			float les_gains__2 = mdl_les_gains(mdl, t0, t1, MODE_MAXIMALE,   2.0, _t_MODE, GRAINE);
			float les_gains__4 = mdl_les_gains(mdl, t0, t1, MODE_MAXIMALE,   4.0, _t_MODE, GRAINE);
			//
			printf("%3.i/%3.i| perf=%f%%", i, I, 100*__pred);
			printf(" score=\033[93m%+f\033[0m (%%.potentiel^2=%+f, %%.potentiel^4=%+f)\n",
				_score,
				les_gains__2,
				les_gains__4
			);
			if (fabs(_score) < 0.00001) {
				printf("Score < 0.00001 => Fin d'optimisation\n");
				break;
			}
		}
	}

	//	Liberer
	if      (methode == SGD)     opti_classe.sgd = 0;
	else if (methode == RMSPROP) liberer_rmsprop(opti_classe.rmsprop);
	else if (methode == ADAM)    liberer_adam   (opti_classe.adam   );
};

void optimiser(
	Mdl_t * mdl,
	uint t0, uint t1,
	float * alpha, float div,
	uint methode, uint I,
	float * pourcent_masque,
	float * pourcent_opti_masque,
	uint PERTURBATIONS,
	uint zero_accumulation_tous_les[C],
	uint _t_MODE, uint GRAINE)
{
	Masque_t * masque = cree_masque(mdl, pourcent_masque, pourcent_opti_masque);
	//
	__interne_optimiser(
		mdl,
		t0, t1,
		alpha, div,
		methode, I,
		masque->masque,
		masque->masque_opti,
		PERTURBATIONS,
		zero_accumulation_tous_les,
		_t_MODE, GRAINE);
	//
	sortire_masque(mdl, masque);
};