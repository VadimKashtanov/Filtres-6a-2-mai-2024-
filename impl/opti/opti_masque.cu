#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define PSEUDO_ALEA_flt(x) (float)((1234*x + 4563) % 1000)/1000.0

#define MASQUE_POIDS   0 	//Academique - DropConnect
#define MASQUE_NEURONE 1 	//Academique - DropOut

#define TYPE_MASQUE MASQUE_POIDS

static __global__ void kerd_cree_masque(
	uint graine,
	uint * masque,
	uint POIDS, float p)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	if (thx < POIDS) {

		masque[thx] = (PSEUDO_ALEA_flt(graine+thx) >= p ? NON_MASQUEE : MASQUEE);
		//	0 - pas masque
		//	1 - masque

		//	masque = ne sera pas optimise
	}
};

static uint ** __cree_masque(
	Mdl_t * mdl, float * p)
{
	uint ** masque = alloc<uint*>(C);
	//
	masque[0] = cudalloc<uint>(mdl->Y[0]);
	kerd_cree_masque<<<dim3(KERD(mdl->Y[0],1024)), dim3(1024)>>>(
		rand() % 10000,
		masque[0],
		mdl->Y[0], p[0]
	);
	ATTENDRE_CUDA();
	//
	FOR (1, c, C) {
		uint POIDS = mdl->inst_POIDS[c];
		masque[c] = cudalloc<uint>(POIDS);
		kerd_cree_masque<<<dim3(KERD(POIDS,1024)), dim3(1024)>>>(
			rand() % 10000,
			masque[c],
			POIDS, p[c]
		);
		ATTENDRE_CUDA();
	};
	return masque;
};

static void __liberer_masque(uint ** masque, uint ** masque_opti) {
	FOR(0, c, C) {
		cudafree<uint>(masque[c]);
		cudafree<uint>(masque_opti[c]);
	}
	free(masque);
	free(masque_opti);
}

// ---------------------------------------------------------

static __global__ void cree_poids_masquees__kerd(
	float * a_masquer, float * origine, uint * masque, uint POIDS)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS)
	{
		if (masque[thx] == MASQUEE) {
			a_masquer[thx] = 0;
		} else {
			a_masquer[thx] = origine[thx];
		}
	};
}

static float ** cree_poids_masques(
	Mdl_t * mdl, uint ** masque)
{
	float ** poids_masques = alloc<float*>(C);
	FOR(1, c, C) {
		uint POIDS = mdl->inst_POIDS[c];
		poids_masques[c] = cudalloc<float>(POIDS);
		cree_poids_masquees__kerd<<<dim3(KERD(POIDS,1024)),dim3(1024)>>>(
			poids_masques[c], mdl->p__d[c], masque[c], POIDS
		);
		ATTENDRE_CUDA();
	}
	return poids_masques;
};

static __global__ void coller_les_poids_masques_kerd(
	float * poids_masques, uint * masque,
	float * anciens_non_masques, uint POIDS)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS)
	{
		if (masque[thx] == MASQUEE) {
			//	poids_masques[thx] normalement = 0
			poids_masques[thx] = anciens_non_masques[thx];
		}
	};
};

static void coller_les_poids_masques(
	Mdl_t * mdl, uint ** masque, float ** non_masques)
{
	FOR(1, c, C) {
		uint POIDS = mdl->inst_POIDS[c];
		coller_les_poids_masques_kerd<<<dim3(KERD(POIDS,1024)),dim3(1024)>>>(
			mdl->p__d[c], masque[c], non_masques[c], POIDS
		);
		ATTENDRE_CUDA();
	}
};

static void liberer_poids_non_masques(float ** poids_non_masques) {
	FOR(1, c, C) CONTROLE_CUDA(hipFree(poids_non_masques[c]));
	free(poids_non_masques);
};

//	==============================================================================
//	==============================================================================
//	==============================================================================

Masque_t * cree_masque(Mdl_t * mdl, float * pourcent, float * pourcent_opti) {
	Masque_t * ret = alloc<Masque_t>(1);
	//
	ret->masque = __cree_masque(mdl, pourcent);
	ret->masque_opti = __cree_masque(mdl, pourcent_opti);
	ret->poids_non_masques = alloc<float*>(C);
	FOR(1, c, C) ret->poids_non_masques[c] = mdl->p__d[c];
	ret->poids_masques = cree_poids_masques(mdl, ret->masque);
	
	//	Temporairement remplacer les poids par les nouveaux masques
	FOR(1, c, C) {
		mdl->p__d[c] = ret->poids_masques[c];
	}
	free(ret->poids_masques);	//car mdl a [C]

	return ret;
};

void sortire_masque(Mdl_t * mdl, Masque_t * masque) {
	//	Coller les poids masques depuis la version non masque (car ils sont =0 dans la partie masquee)
	coller_les_poids_masques(mdl, masque->masque, masque->poids_non_masques);

	//	Finalement remplacer l'ancien non masque
	//	par le ne nouveau masque (mis a jour avec toutes les vrais valeurs)

	// rien a faire c'est deja fait

	liberer_poids_non_masques(masque->poids_non_masques);
	__liberer_masque(masque->masque, masque->masque_opti);
	free(masque);
};