#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define adam_beta1 0.9
#define adam_beta2 0.99

static __global__ void kerd_opti_adam(
	float * p, float * dp, float * v, float * s,
	float alpha, uint POIDS, float div,
	uint couche_filtres)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		float _grad = dp[thx] / div;
		float _v = adam_beta1*v[thx] + (1-adam_beta1)*_grad;
		float _s = adam_beta2*s[thx] + (1-adam_beta2)*_grad*_grad;
		//
		float corr_v = _v / (1 - adam_beta1);
		float corr_s = _s / (1 - adam_beta2);
		//
		p[thx] -= (alpha * _grad * corr_v / (sqrtf(corr_s) + 1e-8) + alpha * L2_regularisation * p[thx] * (uint)(couche_filtres != 1));
		v[thx] = _v;
		s[thx] = _s;
	}
};

static __global__ void kerd_opti_adam_masque(
	float * p, float * dp, float * v, float * s,
	float alpha, uint POIDS, float div, uint * masque, uint * masque_opti,
	uint couche_filtres)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		if (masque[thx] == NON_MASQUEE && masque_opti[thx] == NON_MASQUEE) {
			float _grad = dp[thx] / div;
			float _v = adam_beta1*v[thx] + (1-adam_beta1)*_grad;
			float _s = adam_beta2*s[thx] + (1-adam_beta2)*_grad*_grad;
			//
			float corr_v = _v / (1 - adam_beta1);
			float corr_s = _s / (1 - adam_beta2);
			//
			p[thx] -= (alpha * _grad * corr_v / (sqrtf(corr_s) + 1e-8) + alpha * L2_regularisation * p[thx] * (uint)(couche_filtres != 1));
			v[thx] = _v;
			s[thx] = _s;
		}
	}
};

Adam_t * cree_adam(
	Mdl_t * mdl)
{
	Adam_t * ret = alloc<Adam_t>(1);
	FOR(0, c, C) ret->v[c] = cudazero<float>(mdl->inst_POIDS[c]);
	FOR(0, c, C) ret->s[c] = cudazero<float>(mdl->inst_POIDS[c]);
	return ret;
};

void liberer_adam(Adam_t * adam) {
	FOR(0, c, C) cudafree<float>(adam->v[c]);
	FOR(0, c, C) cudafree<float>(adam->s[c]);
	free(adam);
};

void opti_adam(
	uint zero_accumulation_tous_les[C], uint optimiser_la_couche[C], Mdl_t * mdl, Adam_t * adam,
	float * alpha, float div, uint ** masque, uint ** masque_opti)
{
#define coef_div zero_accumulation_tous_les
	//	Filtres
	if (optimiser_la_couche[0] == 1) {
		uint FILTRES = mdl->inst_POIDS[0];	//pas de *N, car c'est le filtre qu'on ignore, pas les points
		kerd_opti_adam<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
			mdl->p__d[0], mdl->dp__d[0], adam->v[0], adam->s[0], alpha[0], FILTRES, div * coef_div[0],
			(uint)true);
	}
	//	Poids
	FOR(1, c, C) {
		if (optimiser_la_couche[c] == 0) continue;
		
		uint POIDS = mdl->inst_POIDS[c];
		
		if (masque == 0) {
			kerd_opti_adam<<<dim3(KERD(POIDS, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], adam->v[c], adam->s[c],
				alpha[c], POIDS, div * coef_div[c],
				(uint)false
			);
		} else {
			kerd_opti_adam_masque<<<dim3(KERD(POIDS, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], adam->v[c], adam->s[c],
				alpha[c], POIDS, div * coef_div[c], masque[c], masque_opti[c],
				(uint)false
			);
		}
	};
	ATTENDRE_CUDA();
	mdl_poids_gpu_vers_cpu(mdl);
};