#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define rms_alpha 0.90

static __global__ void kerd_opti_rmsprop(
	float * p, float * dp, float * g,
	float alpha, uint POIDS, float div,
	uint couche)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		float _grad = dp[thx] / div;
		float _g = rms_alpha*g[thx] + (1-rms_alpha)*_grad*_grad;
		p[thx] -= (alpha * _grad / (sqrtf(_g) + 1e-8) + alpha * L2_regularisation * p[thx] * (uint)(couche != 0));
		g[thx] = _g;
	}
};

static __global__ void kerd_opti_rmsprop_masque(
	float * p, float * dp, float * g,
	float alpha, uint POIDS, float div, uint * masque, uint * masque_opti,
	uint couche)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		if (masque[thx] == NON_MASQUEE && masque_opti[thx] == NON_MASQUEE) {
			float _grad = dp[thx] / div;
			float _g = rms_alpha*g[thx] + (1-rms_alpha)*_grad*_grad;
			p[thx] -= (alpha * _grad / (sqrtf(_g) + 1e-8) + alpha * L2_regularisation * p[thx] * (uint)(couche != 0));
			g[thx] = _g;
		}
	}
};

Rmsprop_t * cree_rmsprop(
	Mdl_t * mdl)
{
	Rmsprop_t * ret = alloc<Rmsprop_t>(1);
	FOR(0, c, C) ret->g[c] = cudazero<float>(mdl->inst_POIDS[c]);
	return ret;
};

void liberer_rmsprop(Rmsprop_t * rmsprop) {
	FOR(0, c, C) cudafree<float>(rmsprop->g[c]);
	free(rmsprop);
};

void opti_rmsprop(
	uint zero_accumulation_tous_les[C], uint optimiser_la_couche[C], Mdl_t * mdl, Rmsprop_t * rmsprop,
	float * alpha, float div, uint ** masque, uint ** masque_opti)
{
#define coef_div zero_accumulation_tous_les
	//	Filtres
	if (optimiser_la_couche[0] == 1) {
		uint FILTRES = mdl->inst_POIDS[0];	//pas de *N, car c'est le filtre qu'on ignore, pas les points
		kerd_opti_rmsprop<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
			mdl->p__d[0], mdl->dp__d[0], rmsprop->g[0], alpha[0], FILTRES, div * coef_div[0],
			0);
	}
	//	Poids
	FOR(1, c, C) {
		if (optimiser_la_couche[c] == 0) continue;

		uint POIDS = mdl->inst_POIDS[c];
		
		if (masque == 0) {
			kerd_opti_rmsprop<<<dim3(KERD(POIDS, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], rmsprop->g[c],
				alpha[c], POIDS, div * coef_div[c],
				c
			);
		} else {
			kerd_opti_rmsprop_masque<<<dim3(KERD(POIDS, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], rmsprop->g[c],
				alpha[c], POIDS, div * coef_div[c], masque[c], masque_opti[c],
				c
			);
		}
	};
	ATTENDRE_CUDA();
	mdl_poids_gpu_vers_cpu(mdl);
};