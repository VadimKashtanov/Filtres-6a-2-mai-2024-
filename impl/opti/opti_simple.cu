#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

static __global__ void kerd_opti_simple(
	float * p, float * dp, float alpha, uint POIDS, float div,
	uint couche)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		p[thx] -= (alpha * dp[thx] / div + alpha * L2_regularisation * p[thx] * (uint)(couche != 0));
	}
};

static __global__ void kerd_opti_simple_masque(
	float * p, float * dp, float alpha, uint POIDS, float div, uint * masque, uint * masque_opti,
	uint couche)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		if (masque[thx] == NON_MASQUEE && masque_opti[thx] == NON_MASQUEE)
			p[thx] -= (alpha * dp[thx] / div + alpha * L2_regularisation * p[thx] * (uint)(couche != 0));
	}
};

void opti_simple(uint zero_accumulation_tous_les[C], uint optimiser_la_couche[C], Mdl_t * mdl, float * alpha, float div, uint ** masque, uint ** masque_opti) {
#define coef_div zero_accumulation_tous_les
	//	Filtres
	if (optimiser_la_couche[0] == 1) {
		uint FILTRES = mdl->Y[0];	//pas de *N, car c'est le filtre qu'on ignore, pas les points
		if (masque == 0) {
			kerd_opti_simple<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
				mdl->p__d[0], mdl->dp__d[0], alpha[0], FILTRES, div * coef_div[0],
				0);
		} else {
			kerd_opti_simple_masque<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
				mdl->p__d[0], mdl->dp__d[0], alpha[0], FILTRES, div * coef_div[0], masque[0], masque_opti[0],
				0
			);
		}
	}
	//	Poids
	FOR(1, c, C) {
		if (optimiser_la_couche[c] == 0) continue;

		uint POIDS = mdl->inst_POIDS[c];
		if (masque == 0) {
			kerd_opti_simple<<<dim3(KERD(POIDS, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], alpha[c], POIDS, div * coef_div[c],
				c
			);
		} else {
			kerd_opti_simple_masque<<<dim3(KERD(POIDS, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], alpha[c], POIDS, div * coef_div[c], masque[c], masque_opti[c],
				c
			);
		}
	};
	ATTENDRE_CUDA();
	mdl_poids_gpu_vers_cpu(mdl);
};