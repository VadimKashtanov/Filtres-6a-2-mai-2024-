#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

#include "marchee.cuh"

#include "filtres_prixs.cuh"
#include "dot1d_tanh.cuh"
#include "lstm1d_peephole.cuh"
#include "dot1d_tanh_elman.cuh"

static float filtre(
	uint depart,
	float * x, float * f,
	uint intervalle,
	uint type_norme, float _min, float _max)
{
	float normer_x[N];
	//
	FOR(0, i, N) normer_x[i] = x[depart - (i)*intervalle];
	//
	if (type_norme == NORME_CLASSIQUE) {
		_min=normer_x[0];
		_max=normer_x[0];
		//
		FOR(1, i, N) {
			float a = normer_x[i];
			if (a > _max) _max = a;
			if (a < _min) _min = a;
		}
	} else if (type_norme == NORME_THEORIQUE) {
		// rien
	} else if (type_norme == NORME_RELATIVE) {
		_max=fabs(normer_x[0]);
		//
		FOR(1, i, N) {
			float a = fabs(normer_x[i]);
			if (a > _max) _max = a;
		}
		_max = +_max;
		_min = -_max;
	} else {
		ERR("type_norme == %i", type_norme);
	}
	//
	FOR(0, i, N) normer_x[i] = (normer_x[i]-_min)/(_max-_min);
	
	float s = 0, d = 0;
	float f_nouveau = f[0];
	float x_nouveau = normer_x[0];
	//
	float Ps = (0.5+0/N*0.5);
	s += powf(1 + fabs(x_nouveau - f_nouveau), Ps);
	//
	float f_avant = f_nouveau;
	float x_avant = x_nouveau;
	FOR(1, i, N) {
		f_nouveau = f[i];
		x_nouveau = normer_x[i];
		//
		float Ps = (0.5+i/N*0.5);
		float Pd = (1.0+i/N*1.0);
		//
		s += powf(1 + fabs(  x_nouveau   -   f_nouveau  ), Ps);
		d += powf(1 + fabs((x_nouveau-x_avant) - (f_nouveau-f_avant)), Pd);
		f_avant   = f_nouveau;
		x_avant   = x_nouveau;
	};
	
	s = (s/(float)N) - 1;
	d = (d/(float)(N-1))-1;

	return fp_normalisation(expf(-2*s*s-2*d*d));
};

int main(int argc, char ** argv) {
	srand(0);
	//
	FILE * fp = fopen(argv[1], "rb");
	//
	uint * inst_Y = lire<uint>(fp, C);
	uint * _VARS  = lire<uint>(fp, C);
	uint * insts  = lire<uint>(fp, C);
	//
	//
	uint PRIXS_bitget  = lire_un<uint>(fp);
	//
	uint * intervalles = lire<uint>(fp, BLOQUES);
	//
	uint * type_norme  = lire<uint>(fp, BLOQUES);
	uint * _min        = lire<uint>(fp, BLOQUES);
	uint * _max        = lire<uint>(fp, BLOQUES);
	//
	float * lignes     = lire<float>(fp, PRIXS_bitget*BLOQUES);
	//
	float * poids[C];
	FOR(0, c, C) {
		uint POIDS = lire_un<uint>(fp);
		poids[c]   = lire<float>(fp, POIDS);
	}
	//
	fclose(fp);

	//	------------- Espace ----------------
	uint T = (PRIXS_bitget-DEPART);
	//
	float * espace_y[C];
	FOR(0, c, C) {
		espace_y[c] = alloc<float>(T * _VARS[c]);
		memset(espace_y[c], 0, sizeof(float) * T * _VARS[c]);
	}

	uint DEPART_X[C];
	FOR(0, c, C) {
		DEPART_X[c] = _VARS[c] - inst_Y[c];
		//printf("Depart_x = %i\n", DEPART_X[c]);
	}

	//	------------- Calcule ----------------
	FOR(0, t, T) {
		FOR(0, c, C) {
			if (insts[c] == FILTRES_PRIXS) {
				ASSERT(c == 0);
				//
				float * x = NULL;
				float * y = espace_y[0];
				//
				FOR(0, f, BLOQUES*F_PAR_BLOQUES) {
					uint b = (f - (f % F_PAR_BLOQUES)) / F_PAR_BLOQUES;
					y[t*_VARS[c] + f] = filtre(
						b*PRIXS_bitget + DEPART + t,	//depart
						lignes,
						poids[0] + f*N,
						intervalles[b],
						type_norme[b],
						_min[b], _max[b]
					);
				};
			} else if (insts[c] == DOT1D_TANH) {
				float * x = espace_y[c-1];
				float * y = espace_y[ c ];
				//
				uint X = inst_Y[c-1];
				uint Y = inst_Y[ c ];
				//
				FOR(0, i, Y) {
					float s = poids[c][(X+1)*i + X-1+1];
					FOR(0, j, X) {
						s += poids[c][(X+1)*i + j] * x[t*_VARS[c-1] + DEPART_X[c-1] + j];
					}
					y[t*_VARS[c] + i] = dot1d_tanh_ACTIV(i,s);
				};
			} else if (insts[c] == DOT1D_TANH_ELMAN) {
#define CYCLE_DOT1D_TANH_ELMAN 24
				uint condition = (t>0 && !(t % CYCLE_DOT1D_TANH_ELMAN==0));
				float * x = espace_y[c-1];
				float * y = espace_y[ c ];
				//
				uint X = inst_Y[c-1];
				uint Y = inst_Y[ c ];
				//
				FOR(0, _y, Y) {
					float s = poids[c][elman_depart_poid_Ub(X,Y) + _y];
					FOR(0, ix, X) {
						s += poids[c][elman_depart_poid_Ux(X,Y) + _y*X + ix] * x[t*_VARS[c-1] + DEPART_X[c-1] + ix];
					}
					if (condition) {
						FOR(0, iy, Y) {
							s += condition * y[(t-1)*Y + iy] * poids[c][elman_depart_poid_Uy(X,Y) + _y*Y + iy];
						}
					}
					y[t*_VARS[c] + _y] = tanh(s);
				};
			} else if (insts[c] == LSTM1D_PEEPHOLE) {
#define CYCLE_LSTM1D_PEEPHOLE 12
				uint condition = (t>0 && !(t % CYCLE_LSTM1D_PEEPHOLE==0));
				/*
	//	--- Partie fiu ---
	f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
	i = logistique(sI = Ix@x + Ih@h + Ic@c[-1] + Ib)
	u =       tanh(sU = Ux@x + Uh@h +          + Ub)
	//	--- Partie cch ---
	c = f*c[-1] + i*u
	ch = tanh(c)
	//	--- Partie o ---
	o = logistique(sO = Ox@x + Oh@h + Oc@c    + Ob)
	//	--- Partie h ---
	h = o * ch
				*/
				//
				float * x = espace_y[c-1];
				float * y = espace_y[ c ];
				//
				uint X = inst_Y[c-1];
				uint Y = inst_Y[ c ];
				//
				float f[Y], i[Y], u[Y], o[Y];
				//
				memset(f, 0, sizeof(float)*Y);
				memset(i, 0, sizeof(float)*Y);
				memset(u, 0, sizeof(float)*Y);
				memset(o, 0, sizeof(float)*Y);
				//
				FOR(0, _y, Y) {
					//	@x
					FOR(0, ix, X) {
						float val_x = x[t*_VARS[c-1] + DEPART_X[c-1] + ix];
						f[_y] += val_x * poids[c][depart_poids_f(X,Y)+(_y*X+ix)];
						i[_y] += val_x * poids[c][depart_poids_i(X,Y)+(_y*X+ix)];
						u[_y] += val_x * poids[c][depart_poids_u(X,Y)+(_y*X+ix)];
						o[_y] += val_x * poids[c][depart_poids_o(X,Y)+(_y*X+ix)];
					};
						
					//	@h
					FOR(0, ih, Y) {
						float val_h = (condition ? y[(t-1)*_VARS[c] + depart_h(X,Y) + ih] : 0.0);
						f[_y] += val_h * poids[c][depart_poids_f(X,Y)+Fx(X,Y)+(_y*Y+ih)];
						i[_y] += val_h * poids[c][depart_poids_i(X,Y)+Ix(X,Y)+(_y*Y+ih)];
						u[_y] += val_h * poids[c][depart_poids_u(X,Y)+Ux(X,Y)+(_y*Y+ih)];
						o[_y] += val_h * poids[c][depart_poids_o(X,Y)+Ox(X,Y)+(_y*Y+ih)];
					};
						
					//	@c[-1]
					FOR(0, ic, Y) {
						float val_c = (condition ? y[(t-1)*_VARS[c] + depart_c(X,Y) + ic] : 0.0);
						f[_y] += val_c * poids[c][depart_poids_f(X,Y)+Fx(X,Y)+Fh(X,Y)+(_y*Y+ic)];
						i[_y] += val_c * poids[c][depart_poids_i(X,Y)+Ix(X,Y)+Ih(X,Y)+(_y*Y+ic)];
					}
						
					//	+b
					f[_y] += poids[c][depart_poids_f(X,Y)+Fx(X,Y)+Fh(X,Y)+Fc(X,Y)+_y];
					i[_y] += poids[c][depart_poids_i(X,Y)+Ix(X,Y)+Ih(X,Y)+Ic(X,Y)+_y];
					u[_y] += poids[c][depart_poids_u(X,Y)+Ux(X,Y)+Uh(X,Y)        +_y];
					o[_y] += poids[c][depart_poids_o(X,Y)+Ox(X,Y)+Oc(X,Y)+Oc(X,Y)+_y];

					//	()
					f[_y] = logistic(f[_y]);
					i[_y] = logistic(i[_y]);
					u[_y] =     tanh(u[_y]);

					//c = f*c[-1] + i*u
					float c1 = (condition ? y[(t-1)*_VARS[c] + depart_c(X,Y) + _y] : 0.0);
					y[t*_VARS[c] + depart_c(X,Y)  + _y] = f[_y]*c1 + i[_y]*u[_y];
					y[t*_VARS[c] + depart_ch(X,Y) + _y] = lstmpeephole_activ_CH(y[t*_VARS[c] + depart_c(X,Y) + _y]);
				}

				FOR(0, _y, Y) {
					//	+Oc@c
					FOR(0, ic, Y) {
						float val_c = y[t*_VARS[c] + depart_c(X,Y) + ic];
						o[_y] += val_c * poids[c][depart_poids_o(X,Y)+Ox(X,Y)+Oh(X,Y)+(_y*Y+ic)];
					}

					//	()
					o[_y] = logistic(o[_y]);

					//	h
					y[t*_VARS[c] + depart_h(X,Y) + _y] = o[_y] * y[t*_VARS[c] + depart_ch(X,Y) + _y];
				}
			} else {
				ERR("Inst = %i", insts[c]);
			}
		}
	};

	//	---------- Ecrire Resultat ----------
	fp = fopen(argv[1], "wb");
	//
	float res[T];
	FOR(0, t, T) {
		res[t] = espace_y[C-1][t*_VARS[C-1] + DEPART_X[C-1] + 0];
	}
	FWRITE(res, sizeof(float), T, fp);
	//
	fclose(fp);
}