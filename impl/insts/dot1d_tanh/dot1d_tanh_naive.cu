#include "hip/hip_runtime.h"
#include "dot1d_tanh.cuh"

#define BLOQUE_T 16//32
#define BLOQUE_Y 16//32

static __global__ void kerd_dot1d_tanh_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		float s = p[_y*(X+1) + (X+1-1)];
		FOR(0, i, X) s += x[(mega_t*T+0+_t)*X_vars + DEPART_x + i] * p[_y*(X+1) + i];
		float a = dot1d_tanh_ACTIV(_y, s);
		y[/*(depart+_t)*/(mega_t*T+0+_t)*Y + _y] = a;
		locd[/*(depart+_t)*/(mega_t*T+0+_t)*Y + _y] = dot1d_tanh_dACTIV(_y, s,a);
	}
};

void nvidia_dot1d_tanh_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	kerd_dot1d_tanh_naive<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
}

//	============================= Derivation ==============================

static __global__ void kerd_deriv_dot1d_tanh_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		float _locd = locd[/*(depart+_t)*/(mega_t*T+0+_t)*Y + _y] * dy[/*(depart+_t)*/(mega_t*T+0+_t)*Y + _y];
		atomicAdd(&dp[_y*(X+1) + (X+1-1)], _locd);
		FOR(0, i, X) {
			atomicAdd(&dx[(mega_t*T+0+_t)*X_vars + DEPART_x + i], _locd * p[_y*(X+1) + i]);
			atomicAdd(&dp[_y*(X+1) + i], _locd * x[(mega_t*T+0+_t)*X_vars + DEPART_x + i]);
		}
	}
};

void d_nvidia_dot1d_tanh_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	kerd_deriv_dot1d_tanh_naive<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
};