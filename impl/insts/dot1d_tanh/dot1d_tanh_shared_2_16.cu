#include "hip/hip_runtime.h"
#include "dot1d_tanh.cuh"

/*	Difference :
	Au lieux de directement deriver avec que des atomicAdd le
__shared__ noyau, on fait la méthode que j'avais avant
ou on fait une autre opération pour calc dx et dp.

	Mathématiquement ca correspond a deriver y=X@P+B
en dX=p@dY.T
	dx = (p @ ((y-_y)*dtanh(x@p)).T).T
	dp = x.T @ ((y-_y)*dtanh(x@p))
*/

#define BLOQUE     16
#define BLOQUE_MAX 16

static __global__ void kerd_stricte_16__shared2(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, X/BLOQUE) {
		__partage__x[thy][thx] = x[(mega_t*T+0+_t)*( X_vars ) + DEPART_x +d*BLOQUE + thx];
		__partage__p[thy][thx] = p[_y*(X+1) + d*BLOQUE + thy];
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

#define __partage__b __partage__x[0]

	if (thy == 0) __partage__b[thx] = p[_y*(X+1) + (X+1-1)];
	__syncthreads();

	s = (s + __partage__b[thx]);
	float a = dot1d_tanh_ACTIV(_y, s);
	   y[(mega_t*T+0+_t)*Y + _y] = a;
	locd[(mega_t*T+0+_t)*Y + _y] = dot1d_tanh_dACTIV(_y, s,a);
};

void nvidia_dot1d_tanh_shared_2_16(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	if (T%BLOQUE!=0) ERR("ATTENTION T%%16 != 0 (T=%i)", T);
	if (X%BLOQUE==0 && Y%BLOQUE==0 && T%BLOQUE==0) {
		kerd_stricte_16__shared2<<<dim3(KERD(Y, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd);
		ATTENDRE_CUDA();
	} else {
		nvidia_dot1d_tanh_naive(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd);
	}
}

static __global__ void kerd_stricte_16__shared2____dx(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{

	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	//uint _y = thx + blockIdx.x * blockDim.x;
	uint _x = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, Y/BLOQUE) {
		__partage__x[thy][thx] = locd[(mega_t*T+0+_t)*Y+d*BLOQUE+thx] * dy[(mega_t*T+0+_t)*Y+d*BLOQUE+thx];//x[(depart+_t)*( X ) + d*BLOQUE + thx];
		__partage__p[thy][thx] = p[(d*BLOQUE+thy)*(X+1) + _x];//p[_y*(X+1) + d*BLOQUE + thy];
		__syncthreads();

	#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

	dx[(mega_t*T+0+_t)*X_vars+DEPART_x +_x] = s;
};


static __global__ void kerd_stricte_32__shared2____dp(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{

	//dp = x.T @ ((y-_y)*dtanh(x@p))

	__shared__ float __partage__x[BLOQUE_MAX][BLOQUE_MAX];
	__shared__ float __partage__p[BLOQUE_MAX][BLOQUE_MAX];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _x = thx + blockIdx.x * blockDim.x;
	uint _y = thy + blockIdx.y * blockDim.y;

	float s = 0;
	float biais = 0;

	uint d = blockIdx.z;
	//FOR(0, d, T/BLOQUE) {
		__partage__x[thy][thx] = locd[(mega_t*T+0+d*BLOQUE_MAX+thx)*Y+_y] * dy[(mega_t*T+0+d*BLOQUE_MAX+thx)*Y+_y];
		__partage__p[thy][thx] = x[(mega_t*T+0+(d*BLOQUE_MAX+thy))*X_vars+DEPART_x +_x];
		__syncthreads();

	#pragma unroll
		FOR(0, i, BLOQUE_MAX) {
			s += __partage__x[thy][i] * __partage__p[i][thx];
			if (_x == 0) biais += __partage__x[thy][i];
		}
		__syncthreads();
	//};

#define __partage__b __partage__x[0]

	//if (thy == 0) __partage__b[thx] = p[_y*(X+1) + (X+1-1)];
	if (_x == 0) atomicAdd(&dp[_y*(X+1) + (X+1-1)], biais);
	__syncthreads();

	atomicAdd(&dp[_y*(X+1)+_x], s);
};

void d_nvidia_dot1d_tanh_shared_2_16(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	if (T%(MAX2(BLOQUE_MAX,BLOQUE))!=0) ERR("ATTENTION T%%%i != 0 (T=%i)", T, (MAX2(BLOQUE_MAX,BLOQUE)));
	if (X%BLOQUE==0 && Y%BLOQUE==0 && T%BLOQUE==0) {
		kerd_stricte_16__shared2____dx<<<dim3(KERD(X, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
		kerd_stricte_32__shared2____dp<<<dim3(KERD(X, BLOQUE_MAX), KERD(Y, BLOQUE_MAX), DIV(T,BLOQUE_MAX)), dim3(BLOQUE_MAX, BLOQUE_MAX,1)>>>(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
		ATTENDRE_CUDA();
	} else {
		d_nvidia_dot1d_tanh_naive(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
	}
}