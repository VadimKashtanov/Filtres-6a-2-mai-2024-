#include "hip/hip_runtime.h"
#include "dot1d_tanh_elman.cuh"

#define BLOQUE_T 16//32
#define BLOQUE_Y 16//32

static __global__ void kerd_dot1d_tanh_elman_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		float s = 0;
		//
		FOR(0, ix, X) {
			s += x[(mega_t*T+0+_t)*X_vars + DEPART_x + ix] * p[elman_depart_poid_Ux(X,Y) + _y*X + ix];
		}
		if (mega_t != 0) {
			FOR(0, iy, Y) {
				s += y[( (mega_t-1)*T+0+_t )*Y + iy] * p[elman_depart_poid_Uy(X,Y) + _y*Y + iy];
			}
		}
		s += p[elman_depart_poid_Ub(X,Y) + _y];
		//
		float a = tanh(s);
		   y[(mega_t*T+0+_t)*Y + _y] = a;
		locd[(mega_t*T+0+_t)*Y + _y] = d_tanh(s,a);
	}
};

void nvidia_dot1d_tanh_elman_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	kerd_dot1d_tanh_elman_naive<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
}

//	============================= Derivation ==============================

static __global__ void kerd_deriv_dot1d_tanh_elman_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		float _locd = locd[(mega_t*T+0+_t)*Y + _y] * dy[(mega_t*T+0+_t)*Y + _y];
		atomicAdd(&dp[elman_depart_poid_Ub(X,Y) + _y], _locd);
		//
		FOR(0, ix, X) {
			//s += x[(mega_t*T+0+_t)*X_vars + DEPART_x + ix] * p[elman_depart_poid_Ux(X,Y) + _y*X + ix];
			atomicAdd(&dx[(mega_t*T+0+_t)*X_vars + DEPART_x + ix], _locd * p[elman_depart_poid_Ux(X,Y) + _y*X + ix]);
			atomicAdd(&dp[elman_depart_poid_Ux(X,Y) + _y*X + ix], _locd * x[(mega_t*T+0+_t)*X_vars + DEPART_x + ix]);
		}
		if (mega_t != 0) {
			FOR(0, iy, Y) {
				//s += y[( (mega_t-1)*T+0+_t )*Y + iy] * p[elman_depart_poid_Uy(X,Y) + _y*X + iy];
				atomicAdd(&dy[( (mega_t-1)*T+0+_t )*Y + iy], _locd * p[elman_depart_poid_Uy(X,Y) + _y*Y + iy]);
				atomicAdd(&dp[elman_depart_poid_Uy(X,Y) + _y*Y + iy], _locd * y[( (mega_t-1)*T+0+_t )*Y + iy]);
			}
		}
	}
};

void d_nvidia_dot1d_tanh_elman_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	kerd_deriv_dot1d_tanh_elman_naive<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
};