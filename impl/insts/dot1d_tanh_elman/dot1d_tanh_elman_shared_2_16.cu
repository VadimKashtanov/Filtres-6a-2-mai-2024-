#include "hip/hip_runtime.h"
#include "dot1d_tanh_elman.cuh"

/*	Difference :
	Au lieux de directement deriver avec que des atomicAdd le
__shared__ noyau, on fait la méthode que j'avais avant
ou on fait une autre opération pour calc dx et dp.

	Mathématiquement ca correspond a deriver y=X@P+B
en dX=p@dY.T
	dx = (p @ ((y-_y)*dtanh(x@p)).T).T
	dp = x.T @ ((y-_y)*dtanh(x@p))
*/

#define BLOQUE     16
#define BLOQUE_MAX 16

static __global__ void kerd_elman_stricte_16__shared2(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	//	Ux@x - elman_depart_poid_Ux(X,Y)
	FOR(0, d, X/BLOQUE) {
		__partage__x[thy][thx] = x[(mega_t*T+0+_t)*( X_vars ) + DEPART_x + (d*BLOQUE + thx)];
		__partage__p[thy][thx] = p[elman_depart_poid_Ux(X,Y) + _y*X + (d*BLOQUE + thy)];
		__syncthreads();

		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

	//	Uy@y[-1] - elman_depart_poid_Ux(X,Y)
	if (mega_t != 0) {
		FOR(0, d, Y/BLOQUE) {
			__partage__x[thy][thx] = y[((mega_t-1)*T+0+_t)*Y + d*BLOQUE + thx];
			__partage__p[thy][thx] = p[elman_depart_poid_Uy(X,Y) + _y*Y + (d*BLOQUE + thy)];
			__syncthreads();

			FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
			__syncthreads();
		};
	}

	//	+ Ub - elman_depart_poid_Ub(X,Y)
#define __partage__b __partage__x[0]
	if (thy == 0) __partage__b[thx] = p[elman_depart_poid_Ub(X,Y) + _y];
	__syncthreads();

	s = (s + __partage__b[thx]);
	float a = tanh(s);
	   y[(mega_t*T+0+_t)*Y + _y] = a;
	locd[(mega_t*T+0+_t)*Y + _y] = d_tanh(s,a);
};

void nvidia_dot1d_tanh_elman_shared_2_16(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	if (T%BLOQUE!=0) ERR("ATTENTION T%%16 != 0 (T=%i)", T);
	if (X%BLOQUE==0 && Y%BLOQUE==0 && T%BLOQUE==0) {
		kerd_elman_stricte_16__shared2<<<dim3(KERD(Y, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd);
		ATTENDRE_CUDA();
	} else {
		nvidia_dot1d_tanh_elman_naive(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd);
	}
}

//	======================================================================================
//	======================================================================================
//	======================================================================================

static __global__ void kerd_elman_stricte_16__shared2____dx(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{

	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	//uint _y = thx + blockIdx.x * blockDim.x;
	uint _x = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, Y/BLOQUE) {
		__partage__x[thy][thx] = locd[(mega_t*T+0+_t)*Y+d*BLOQUE+thx] * dy[(mega_t*T+0+_t)*Y+d*BLOQUE+thx];
		__partage__p[thy][thx] = p[elman_depart_poid_Ux(X,Y) + (d*BLOQUE+thy)*X + _x];
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

	atomicAdd(&dx[(mega_t*T+0+_t)*X_vars+DEPART_x +_x], s);
};

static __global__ void kerd_elman_stricte_16__shared2____dy(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{

	//dy = (p @ ((y-_y)*dtanh(p@y)).T).T

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, Y/BLOQUE) {
		__partage__x[thy][thx] = locd[(mega_t*T+0+_t)*Y+d*BLOQUE+thx] * dy[(mega_t*T+0+_t)*Y + (d*BLOQUE+thx)];
		__partage__p[thy][thx] =    p[elman_depart_poid_Uy(X,Y) + (d*BLOQUE+thy)*Y + _y];
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

	atomicAdd(&dy[((mega_t-1)*T+0+_t)*Y + _y], s);
};


static __global__ void kerd_elman_stricte_32__shared2____dUxUb(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{

	//dp = x.T @ ((y-_y)*dtanh(x@p))

	__shared__ float __partage__x[BLOQUE_MAX][BLOQUE_MAX];
	__shared__ float __partage__p[BLOQUE_MAX][BLOQUE_MAX];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _x = thx + blockIdx.x * blockDim.x;
	uint _y = thy + blockIdx.y * blockDim.y;

	float s = 0;
	float biais = 0;

	uint d = blockIdx.z;
	//FOR(0, d, T/BLOQUE) {
		__partage__x[thy][thx] = locd[(mega_t*T+0+d*BLOQUE_MAX+thx)*Y+_y] * dy[(mega_t*T+0+d*BLOQUE_MAX+thx)*Y+_y];
		__partage__p[thy][thx] = x[(mega_t*T+0+(d*BLOQUE_MAX+thy))*X_vars+DEPART_x +_x];
		__syncthreads();

	#pragma unroll
		FOR(0, i, BLOQUE_MAX) {
			s += __partage__x[thy][i] * __partage__p[i][thx];
			if (_x == 0) biais += __partage__x[thy][i];
		}
		__syncthreads();
	//};

#define __partage__b __partage__x[0]

	//if (thy == 0) __partage__b[thx] = p[_y*(X+1) + (X+1-1)];
	if (_x == 0) atomicAdd(&dp[elman_depart_poid_Ub(X,Y) + _y], biais);
	__syncthreads();

	atomicAdd(&dp[elman_depart_poid_Ux(X,Y) + _y*X + _x], s);
};

static __global__ void kerd_elman_stricte_32__shared2____dUy(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{

	//dUy = y[-1].T @ ((y-_y)*dtanh(y[-1]@p))

	__shared__ float __partage__x[BLOQUE_MAX][BLOQUE_MAX];
	__shared__ float __partage__p[BLOQUE_MAX][BLOQUE_MAX];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y1 = thx + blockIdx.x * blockDim.x;	//pour y[-1]
	uint _y  = thy + blockIdx.y * blockDim.y;	//pour y

	float s = 0;

	uint d = blockIdx.z;
	//FOR(0, d, T/BLOQUE) {
		__partage__x[thy][thx] = locd[( mega_t   *T+0+(d*BLOQUE_MAX+thx))*Y + _y ] * dy[(mega_t*T+0+d*BLOQUE_MAX+thx)*Y+_y];
		__partage__p[thy][thx] =    y[((mega_t-1)*T+0+(d*BLOQUE_MAX+thy))*Y + _y1];
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE_MAX) {
			s += __partage__x[thy][i] * __partage__p[i][thx];
		}
		__syncthreads();
	//};
	atomicAdd(&dp[elman_depart_poid_Uy(X,Y) + _y*Y + _y1], s);
};

void d_nvidia_dot1d_tanh_elman_shared_2_16(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	if (T%(MAX2(BLOQUE_MAX,BLOQUE))!=0) ERR("ATTENTION T%%%i != 0 (T=%i)", T, (MAX2(BLOQUE_MAX,BLOQUE)));
	if (X%BLOQUE==0 && Y%BLOQUE==0 && T%BLOQUE==0) {
		kerd_elman_stricte_16__shared2____dx<<<dim3(KERD(X, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
		kerd_elman_stricte_32__shared2____dUxUb<<<dim3(KERD(X, BLOQUE_MAX), KERD(Y, BLOQUE_MAX), DIV(T,BLOQUE_MAX)), dim3(BLOQUE_MAX, BLOQUE_MAX,1)>>>(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
		if (mega_t != 0) {
			kerd_elman_stricte_32__shared2____dUy<<<dim3(KERD(Y, BLOQUE_MAX), KERD(Y, BLOQUE_MAX), DIV(T,BLOQUE_MAX)), dim3(BLOQUE_MAX, BLOQUE_MAX,1)>>>(
				mega_t,
				_t_MODE, GRAINE,
				X_vars, Y_vars,
				X, Y,
				depart, T,
				DEPART_x,
				x, y,
				p,
				locd,
				dy,
				dx,
				dp);
			kerd_elman_stricte_16__shared2____dy<<<dim3(KERD(Y, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
				mega_t,
				_t_MODE, GRAINE,
				X_vars, Y_vars,
				X, Y,
				depart, T,
				DEPART_x,
				x, y,
				p,
				locd,
				dy,
				dx,
				dp);
		}
		ATTENDRE_CUDA();
	} else {
		d_nvidia_dot1d_tanh_elman_naive(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
	}
}