#include "hip/hip_runtime.h"
#include "dot1d_tanh_elman.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void cree_dot1d_tanh_elman(Mdl_t * mdl, uint c)
{
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	mdl->inst_POIDS        [c] = (X*Y + Y*Y + Y);
	mdl->inst_VARS         [c] = Y;
	mdl->inst_LOCDS        [c] = Y;
	mdl->inst_SORTIES      [c] = Y;
	mdl->inst_DEPART_SORTIE[c] = mdl->inst_VARS[c] - mdl->Y[c];
	//
	mdl->p[c] = alloc<float>(mdl->inst_POIDS[c]);

	FOR(0, i, mdl->inst_POIDS[c]) {
		mdl->p[c][i] = (2*rnd()-1) * sqrtf(/*10.0*/ 8.0 / (X+Y));
	}
	//
	mdl->inst_p_separateurs[c] = 0;
};

void plume_dot1d_tanh_elman(Mdl_t * mdl, uint c)
{
	printf("POIDS dot1d_tanh: \n");
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	TODO();
};

//	=========================================================

void regulariser_dot1d_tanh_elman(Mdl_t * mdl, uint c) {

};

//	=========================================================

void f_dot1d_tanh_elman(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE, uint mega_t) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	//
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	if (mode == MODE_NAIF) {
		nvidia_dot1d_tanh_elman_naive(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else if (mode == MODE_MAXIMALE) {
		nvidia_dot1d_tanh_elman_shared_2_16(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda f(x)", mode);
	}
}

//	----------------------------

void df_dot1d_tanh_elman(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE, uint mega_t) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	//
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	//
	if (mode == MODE_NAIF) {
		d_nvidia_dot1d_tanh_elman_naive(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else if (mode == MODE_MAXIMALE) {
		d_nvidia_dot1d_tanh_elman_shared_2_16(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda df(x)", mode);
	}
}