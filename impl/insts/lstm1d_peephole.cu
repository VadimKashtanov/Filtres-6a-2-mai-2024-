#include "hip/hip_runtime.h"
#include "lstm1d_peephole.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void cree_lstm1d_peephole(Mdl_t * mdl, uint c)
{
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	//
	mdl->inst_POIDS        [c] = lstm_POIDS(X,Y);
	mdl->inst_VARS         [c] = lstm_VARS(X,Y);
	mdl->inst_LOCDS        [c] = lstm_LOCDS(X,Y);
	mdl->inst_SORTIES      [c] = Y;
	mdl->inst_DEPART_SORTIE[c] = mdl->inst_VARS[c] - mdl->inst_SORTIES[c];
	//
	mdl->p[c] = alloc<float>(mdl->inst_POIDS[c]);
	//
	FOR(0, i, mdl->inst_POIDS[c]) {
		mdl->p[c][i] = (2*rnd()-1) * sqrtf(/*10.0*/ 6.0 / (X+Y));
	}
	//
	mdl->inst_p_separateurs[c] = 15;
	mdl->char_inst_p_separateurs[c] = alloc<char*>(mdl->inst_p_separateurs[c]);
	mdl->depart_inst_p_separateurs[c] = alloc<uint>(mdl->inst_p_separateurs[c]);
	//
	//	F
	mdl->char_inst_p_separateurs  [c][0] = "Fx";
	mdl->depart_inst_p_separateurs[c][0] = depart_poids_f(X,Y)+0;
	mdl->char_inst_p_separateurs  [c][1] = "Fh";
	mdl->depart_inst_p_separateurs[c][1] = depart_poids_f(X,Y)+Fx(X,Y);
	mdl->char_inst_p_separateurs  [c][2] = "Fc";
	mdl->depart_inst_p_separateurs[c][2] = depart_poids_f(X,Y)+Fx(X,Y)+Fh(X,Y);
	mdl->char_inst_p_separateurs  [c][3] = "Fb";
	mdl->depart_inst_p_separateurs[c][3] = depart_poids_f(X,Y)+Fx(X,Y)+Fh(X,Y)+Fc(X,Y);
	//	I
	mdl->char_inst_p_separateurs  [c][4] = "Ix";
	mdl->depart_inst_p_separateurs[c][4] = depart_poids_i(X,Y)+0;
	mdl->char_inst_p_separateurs  [c][5] = "Ih";
	mdl->depart_inst_p_separateurs[c][5] = depart_poids_i(X,Y)+Ix(X,Y);
	mdl->char_inst_p_separateurs  [c][6] = "Ic";
	mdl->depart_inst_p_separateurs[c][6] = depart_poids_i(X,Y)+Ix(X,Y)+Ih(X,Y);
	mdl->char_inst_p_separateurs  [c][7] = "Ib";
	mdl->depart_inst_p_separateurs[c][7] = depart_poids_i(X,Y)+Ix(X,Y)+Ih(X,Y)+Ic(X,Y);
	//	U
	mdl->char_inst_p_separateurs  [c][8] = "Ux";
	mdl->depart_inst_p_separateurs[c][8] = depart_poids_u(X,Y)+0;
	mdl->char_inst_p_separateurs  [c][9] = "Uh";
	mdl->depart_inst_p_separateurs[c][9] = depart_poids_u(X,Y)+Ux(X,Y);
	mdl->char_inst_p_separateurs  [c][10] = "Ub";
	mdl->depart_inst_p_separateurs[c][10] = depart_poids_u(X,Y)+Ux(X,Y)+Uh(X,Y);
	//	O
	mdl->char_inst_p_separateurs  [c][11] = "Ox";
	mdl->depart_inst_p_separateurs[c][11] = depart_poids_o(X,Y)+0;
	mdl->char_inst_p_separateurs  [c][12] = "Oh";
	mdl->depart_inst_p_separateurs[c][12] = depart_poids_o(X,Y)+Ix(X,Y);
	mdl->char_inst_p_separateurs  [c][13] = "Oc";
	mdl->depart_inst_p_separateurs[c][13] = depart_poids_o(X,Y)+Ix(X,Y)+Ih(X,Y);
	mdl->char_inst_p_separateurs  [c][14] = "Ob";
	mdl->depart_inst_p_separateurs[c][14] = depart_poids_o(X,Y)+Ix(X,Y)+Ih(X,Y)+Ic(X,Y);
};

void plume_lstm1d_peephole(Mdl_t * mdl, uint c)
{
	printf("POIDS lstm1d_peephole: \n");
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
};

//	=========================================================

static void equilibrer_vecteur(float * p, uint X) {
	float na = 0;
	FOR(0, x, X) na += fabs(p[x]);
	float je_veux = 15.0;
	float coef = je_veux / na;
	FOR(0, x, X) p[x] *= coef;
} 

void regulariser_lstm1d_peephole(Mdl_t * mdl, uint c) {
	/*uint X=mdl->Y[c-1], Y=mdl->Y[c];
	//
	FOR(0, y, Y) {	//Pas les Biais
		//	f
		equilibrer_vecteur(mdl->p[c] + depart_poids_f(X,Y)+y*X,                 X);
		equilibrer_vecteur(mdl->p[c] + depart_poids_f(X,Y)+Fx(X,Y)+y*Y,         Y);
		equilibrer_vecteur(mdl->p[c] + depart_poids_f(X,Y)+Fx(X,Y)+Fh(X,Y)+y*Y, Y);
		//	i
		equilibrer_vecteur(mdl->p[c] + depart_poids_i(X,Y)+y*X,                 X);
		equilibrer_vecteur(mdl->p[c] + depart_poids_i(X,Y)+Ix(X,Y)+y*Y,         Y);
		equilibrer_vecteur(mdl->p[c] + depart_poids_i(X,Y)+Ix(X,Y)+Ih(X,Y)+y*Y, Y);
		//	u
		equilibrer_vecteur(mdl->p[c] + depart_poids_u(X,Y)+y*X,                 X);
		equilibrer_vecteur(mdl->p[c] + depart_poids_u(X,Y)+Ux(X,Y)+y*Y,         Y);
		//	o
		equilibrer_vecteur(mdl->p[c] + depart_poids_o(X,Y)+y*X,                 X);
		equilibrer_vecteur(mdl->p[c] + depart_poids_o(X,Y)+Ox(X,Y)+y*Y,         Y);
		equilibrer_vecteur(mdl->p[c] + depart_poids_o(X,Y)+Ox(X,Y)+Oh(X,Y)+y*Y, Y);
	};*/
};

//	=========================================================

void f_lstm1d_peephole(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE, uint mega_t) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	//
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	//
	if (mode == MODE_NAIF) {
		nvidia_lstm1d_peephole_naive(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else if (mode == MODE_MAXIMALE) {
		nvidia_lstm1d_peephole_shared_16_2(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda f(x)", mode);
	}
}

//	----------------------------

void df_lstm1d_peephole(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE, uint mega_t) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	//
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	//
	if (mode == MODE_NAIF) {
		d_nvidia_lstm1d_peephole_naive(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else if (mode = MODE_MAXIMALE) {
		d_nvidia_lstm1d_peephole_shared_16_2(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda df(x)", mode);
	}
}