#include "hip/hip_runtime.h"
#include "dot1d_tanh.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void cree_dot1d_tanh(Mdl_t * mdl, uint c)
{
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	mdl->inst_POIDS        [c] = (mdl->Y[c-1]+1)*mdl->Y[c];
	mdl->inst_VARS         [c] = mdl->Y[c];
	mdl->inst_LOCDS        [c] = mdl->Y[c];
	mdl->inst_SORTIES      [c] = mdl->Y[c];
	mdl->inst_DEPART_SORTIE[c] = mdl->Y[c] - mdl->Y[c];
	//
	mdl->p[c] = alloc<float>(mdl->inst_POIDS[c]);

	FOR(0, y, Y) {
		FOR(0, x, X+1) {
			mdl->p[c][y*(X+1)+x] = (2*rnd()-1) * sqrtf(/*10.0*/ 8.0 / (X+Y));
		}
	}
	//
	mdl->inst_p_separateurs[c] = 0;
};

void plume_dot1d_tanh(Mdl_t * mdl, uint c)
{
	printf("POIDS dot1d_tanh: \n");
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	FOR(0, y, Y) {
		printf("y=%i : ", y);
		FOR(0, x, X) {
			printf("%+f,", mdl->p[c][y*(X+1)+x]);
		}
		printf(" biais=%+f\n", mdl->p[c][y*(X+1)+X+1-1]);
	}
};

//	=========================================================

void regulariser_dot1d_tanh(Mdl_t * mdl, uint c) {
	/*uint X=mdl->Y[c-1], Y=mdl->Y[c];
	//
	//float na = somme_absolue_horizontale<float>(mdl->p);
	float na;
	FOR(0, y, Y) {
		na = 0;
		FOR(0, x, X+1) na += fabs(mdl->p[c][y*(X+1) + x]);
		float je_veux = 15.0;
		float coef = je_veux / na;
		FOR(0, x, X+1) mdl->p[c][y*(X+1) + x] *= coef;
	}*/
};

//	=========================================================

void f_dot1d_tanh(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE, uint mega_t) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	//
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	if (mode == MODE_NAIF) {
		nvidia_dot1d_tanh_naive(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else if (mode == MODE_MAXIMALE) {
		nvidia_dot1d_tanh_shared_2_16(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda f(x)", mode);
	}
}

//	----------------------------

void df_dot1d_tanh(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE, uint mega_t) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	//
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	//
	if (mode == MODE_NAIF) {
		d_nvidia_dot1d_tanh_naive(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else if (mode == MODE_MAXIMALE) {
		d_nvidia_dot1d_tanh_shared_2_16(
			mega_t,
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, mdl->T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda df(x)", mode);
	}
}