#include "hip/hip_runtime.h"
#include "lstm1d_peephole.cuh"

#define BLOQUE_T 16//32
#define BLOQUE_Y 16//32

/*
//	--- Partie fiu ---
f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
i = logistique(sI = Ix@x + Ih@h + Ic@c[-1] + Ib)
u =       tanh(sU = Ux@x + Uh@h +          + Ub)
//	--- Partie cch ---
c = f*c[-1] + i*u
ch = tanh(c)
//	--- Partie o ---
o = logistique(sO = Ox@x + Oh@h + Oc@c    + Ob)
//	--- Partie h ---
h = o * ch
*/

static __global__ void kerd_lstm1d_peephole_naive___partie_fiu(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	//	--- Partie fiu ---
	//f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
	//i = logistique(sI = Ix@x + Ih@h + Ic@c[-1] + Ib)
	//u =       tanh(sU = Ux@x + Uh@h +          + Ub)

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float sf = 0;
		float si = 0;
		float su = 0;
		//
		FOR(0, i_x, X) {	//x
			float _x = x[(mega_t*T+0+_t)*X_vars + DEPART_x + i_x];
			sf += _x * p[depart_poids_f(X,Y) + _y*X + i_x];
			si += _x * p[depart_poids_i(X,Y) + _y*X + i_x];
			su += _x * p[depart_poids_u(X,Y) + _y*X + i_x];
		};
		FOR(0, i_y, Y) {	//h & c
			float _c = (mega_t==0 ? 0.0 : y[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + i_y]);
			float _h = (mega_t==0 ? 0.0 : y[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_h(X,Y) + i_y]);
			//
			sf += _h * p[depart_poids_f(X,Y) + Fx(X,Y) + _y*Y + i_y];
			si += _h * p[depart_poids_i(X,Y) + Ix(X,Y) + _y*Y + i_y];
			su += _h * p[depart_poids_u(X,Y) + Ux(X,Y) + _y*Y + i_y];
			//
			sf += _c * p[depart_poids_f(X,Y) + Fx(X,Y) + Fh(X,Y) + _y*Y + i_y];
			si += _c * p[depart_poids_i(X,Y) + Ix(X,Y) + Ih(X,Y) + _y*Y + i_y];
			//su += _c * p[depart_poids_u(X,Y) + Ux(X,Y) + Uh(X,Y) + _y*Y + i_y];
		};
		//
		sf += p[depart_poids_f(X,Y) + Fx(X,Y) + Fh(X,Y) + Fc(X,Y) + _y];
		si += p[depart_poids_i(X,Y) + Ix(X,Y) + Ih(X,Y) + Ic(X,Y) + _y];
		su += p[depart_poids_u(X,Y) + Ux(X,Y) + Uh(X,Y) +         + _y];
		//
		float a_sf = logistic(sf);
		float a_si = logistic(si);
		float a_su =     tanh(su);
		//
		y[(mega_t*T+0+_t)*_lstm_VARS + depart_f(X,Y) + _y] = a_sf;
		y[(mega_t*T+0+_t)*_lstm_VARS + depart_i(X,Y) + _y] = a_si;
		y[(mega_t*T+0+_t)*_lstm_VARS + depart_u(X,Y) + _y] = a_su;
		//
		locd[(mega_t*T+0+_t)*_lstm_LOCDS + depart_dsF(X,Y) + _y] = d_logistic(sf,a_sf);
		locd[(mega_t*T+0+_t)*_lstm_LOCDS + depart_dsI(X,Y) + _y] = d_logistic(si,a_si);
		locd[(mega_t*T+0+_t)*_lstm_LOCDS + depart_dsU(X,Y) + _y] =     d_tanh(su,a_su);
	}
};

static __global__ void kerd_lstm1d_peephole_naive___partie_cch(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	//	--- Partie c ---
	//c = f*c[-1] + i*u

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float _f  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_f(X,Y) + _y];
		float _c1 = (mega_t==0 ? 0.0 : y[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + _y]);
		float _i  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_i(X,Y) + _y];
		float _u  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_u(X,Y) + _y];
		//
		float _c = _f*_c1 + _i*_u;
		//
		y[(mega_t*T+0+_t)*_lstm_VARS + depart_c (X,Y) + _y] = _c;
		y[(mega_t*T+0+_t)*_lstm_VARS + depart_ch(X,Y) + _y] = lstmpeephole_activ_CH(_c);
	}
}

static __global__ void kerd_lstm1d_peephole_naive___partie_o(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	//	--- Partie och ---
	//o = logistique(sO = Ox@x + Oh@h + Oc@c + Ob)

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float so = 0;
		//
		FOR(0, i_x, X) {	//x
			float _x = x[(mega_t*T+0+_t)*X_vars + DEPART_x + i_x];
			so += _x * p[depart_poids_o(X,Y) + _y*X + i_x];
		};
		FOR(0, i_y, Y) {	//h & c
			float _c = (mega_t==0 ? 0.0 : y[(mega_t    *T+0+_t)*_lstm_VARS + depart_c(X,Y) + i_y]);
			float _h = (mega_t==0 ? 0.0 : y[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_h(X,Y) + i_y]);
			//
			so += _h * p[depart_poids_o(X,Y) + Ox(X,Y)           + _y*Y + i_y];
			//
			so += _c * p[depart_poids_o(X,Y) + Ox(X,Y) + Oh(X,Y) + _y*Y + i_y];
		};
		//
		so += p[depart_poids_o(X,Y) + Ox(X,Y) + Oh(X,Y) + Oc(X,Y) + _y];
		//
		float a_so = logistic(so);
		//
		   y[(mega_t*T+0+_t)*_lstm_VARS  + depart_o(X,Y)   + _y] = a_so;
		//
		locd[(mega_t*T+0+_t)*_lstm_LOCDS + depart_dsO(X,Y) + _y] = d_logistic(so,a_so);
	}
}

static __global__ void kerd_lstm1d_peephole_naive___partie_h(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	//	--- Partie h ---
	//h = o * ch

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float _o  = y[(mega_t*T+0+_t)*_lstm_VARS + depart_o (X,Y) + _y];
		float _ch = y[(mega_t*T+0+_t)*_lstm_VARS + depart_ch(X,Y) + _y];
		//
		y[(mega_t*T+0+_t)*_lstm_VARS + depart_h(X,Y) + _y] = _o * _ch;
	}
}

void nvidia_lstm1d_peephole_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	//	--- Partie fiu ---
	kerd_lstm1d_peephole_naive___partie_fiu<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
	//	--- Partie c ---
	kerd_lstm1d_peephole_naive___partie_cch<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
	//	--- Partie och ---
	kerd_lstm1d_peephole_naive___partie_o<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
	//	--- Partie h ---
	kerd_lstm1d_peephole_naive___partie_h<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
}