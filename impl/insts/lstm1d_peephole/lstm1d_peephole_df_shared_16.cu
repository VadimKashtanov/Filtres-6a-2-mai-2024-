#include "hip/hip_runtime.h"
#include "lstm1d_peephole.cuh"

#define BLOQUE_T 16
#define BLOQUE_Y 16

/*
//	--- Partie fiu ---
f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
i = logistique(sI = Ix@x + Ih@h + Ic@c[-1] + Ib)
u =       tanh(sU = Ux@x + Uh@h +          + Ub)
//	--- Partie cch ---
c = f*c[-1] + i*u
ch = tanh(c)
//	--- Partie o ---
o = logistique(sO = Ox@x + Oh@h + Oc@c    + Ob)
//	--- Partie h ---
h = o * ch
*/

static void d_lstm_peephole_f(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l,
	float * dy,
	float * dx,
	float * dp)
{
	float * x0=x; uint X0_vars=X_vars;         uint X0=X; uint depart_x0=DEPART_x;      float * dx0=dx; uint x0_depart__t=mega_t*T;
	float * x1=y; uint X1_vars=lstm_VARS(X,Y); uint X1=Y; uint depart_x1=depart_h(X,Y); float * dx1=dy; uint x1_depart__t=(mega_t-1)*T;
	float * x2=y; uint X2_vars=lstm_VARS(X,Y); uint X2=Y; uint depart_x2=depart_c(X,Y); float * dx2=dy; uint x2_depart__t=(mega_t-1)*T;
	//
	uint Y__vars=lstm_VARS (X,Y); uint depart__y=depart_f  (X,Y); uint y__depart__t=mega_t*T;
	uint L__vars=lstm_LOCDS(X,Y); uint depart__l=depart_dsF(X,Y);
	//
	float * a0=p; uint depart_a0=depart_poids_f(X,Y);						 float * da0=dp;
	float * a1=p; uint depart_a1=depart_poids_f(X,Y)+Fx(X,Y);				 float * da1=dp;
	float * a2=p; uint depart_a2=depart_poids_f(X,Y)+Fx(X,Y)+Fh(X,Y);		 float * da2=dp;
	float * b =p; uint depart__b=depart_poids_f(X,Y)+Fx(X,Y)+Fh(X,Y)+Fc(X,Y);float * db =dp;
	//
	uint activation = cuda_math_LOGISTIC;
	//
	if (mega_t == 0) {
		//f = logistique(sF = Fx@x +             + Fb)
		d_nvidia_F_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, dx0, x0_depart__t,
			//
			y, Y__vars, Y, depart__y, dy, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0, da0,
			b , depart__b, db,
			//
			activation);
	} else {
		//f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
		d_nvidia_F_AX_AX_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, dx0, x0_depart__t,
			x1, X1_vars, X1, depart_x1, dx1, x1_depart__t,
			x2, X2_vars, X2, depart_x2, dx2, x2_depart__t,
			//
			y, Y__vars, Y, depart__y, dy, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0, da0,
			a1, depart_a1, da1,
			a2, depart_a2, da2,
			b , depart__b, db,
			//
			activation);
	}
}

static void d_lstm_peephole_i(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l,
	float * dy,
	float * dx,
	float * dp)
{
	float * x0=x; uint X0_vars=X_vars;         uint X0=X; uint depart_x0=DEPART_x;      float * dx0=dx; uint x0_depart__t=mega_t*T;
	float * x1=y; uint X1_vars=lstm_VARS(X,Y); uint X1=Y; uint depart_x1=depart_h(X,Y); float * dx1=dy; uint x1_depart__t=(mega_t-1)*T;
	float * x2=y; uint X2_vars=lstm_VARS(X,Y); uint X2=Y; uint depart_x2=depart_c(X,Y); float * dx2=dy; uint x2_depart__t=(mega_t-1)*T;
	//
	uint Y__vars=lstm_VARS (X,Y); uint depart__y=depart_i  (X,Y); uint y__depart__t=mega_t*T;
	uint L__vars=lstm_LOCDS(X,Y); uint depart__l=depart_dsI(X,Y);
	//
	float * a0=p; uint depart_a0=depart_poids_i(X,Y);						 float * da0=dp;
	float * a1=p; uint depart_a1=depart_poids_i(X,Y)+Ix(X,Y);				 float * da1=dp;
	float * a2=p; uint depart_a2=depart_poids_i(X,Y)+Ix(X,Y)+Ih(X,Y);		 float * da2=dp;
	float * b =p; uint depart__b=depart_poids_i(X,Y)+Ix(X,Y)+Ih(X,Y)+Ic(X,Y);float * db =dp;
	//
	uint activation = cuda_math_LOGISTIC;
	//
	if (mega_t == 0) {
		//f = logistique(sF = Fx@x + Fh@h +          + Fb)
		d_nvidia_F_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, dx0, x0_depart__t,
			//
			y, Y__vars, Y, depart__y, dy, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0, da0,
			b , depart__b, db,
			//
			activation);
	} else {
		//f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
		d_nvidia_F_AX_AX_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, dx0, x0_depart__t,
			x1, X1_vars, X1, depart_x1, dx1, x1_depart__t,
			x2, X2_vars, X2, depart_x2, dx2, x2_depart__t,
			//
			y, Y__vars, Y, depart__y, dy, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0, da0,
			a1, depart_a1, da1,
			a2, depart_a2, da2,
			b , depart__b, db,
			//
			activation);
	}
}

static void d_lstm_peephole_u(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l,
	float * dy,
	float * dx,
	float * dp)
{
	float * x0=x; uint X0_vars=X_vars;         uint X0=X; uint depart_x0=DEPART_x;      float * dx0=dx; uint x0_depart__t=mega_t*T;
	float * x1=y; uint X1_vars=lstm_VARS(X,Y); uint X1=Y; uint depart_x1=depart_h(X,Y); float * dx1=dy; uint x1_depart__t=(mega_t-1)*T;
	//
	uint Y__vars=lstm_VARS (X,Y); uint depart__y=depart_u  (X,Y); uint y__depart__t=mega_t*T;
	uint L__vars=lstm_LOCDS(X,Y); uint depart__l=depart_dsU(X,Y);
	//
	float * a0=p; uint depart_a0=depart_poids_u(X,Y);				 float * da0=dp;
	float * a1=p; uint depart_a1=depart_poids_u(X,Y)+Ux(X,Y);		 float * da1=dp;
	float * b =p; uint depart__b=depart_poids_u(X,Y)+Ux(X,Y)+Uh(X,Y);float * db =dp;
	//
	uint activation = cuda_math_TANH;
	//
	if (mega_t == 0) {
		//f = logistique(sF = Fx@x + Fh@h +          + Fb)
		d_nvidia_F_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, dx0, x0_depart__t,
			//
			y, Y__vars, Y, depart__y, dy, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0, da0,
			b , depart__b, db,
			//
			activation);
	} else {
		//f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
		d_nvidia_F_AX_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, dx0, x0_depart__t,
			x1, X1_vars, X1, depart_x1, dx1, x1_depart__t,
			//
			y, Y__vars, Y, depart__y, dy, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0, da0,
			a1, depart_a1, da1,
			b , depart__b, db,
			//
			activation);
	}
}

static __global__ void d_kerd_lstm1d_peephole_naive___partie_cch(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l,
	float * dy,
	float * dx,
	float * dp)
{
	//	--- Partie c ---
	//c = f*c[-1] + i*u
	//ch = tanh(c)

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float _f  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_f(X,Y) + _y];
		float _c1 = (mega_t==0 ? 0.0 : y[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + _y]);
		float _i  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_i(X,Y) + _y];
		float _u  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_u(X,Y) + _y];
		//
		float _c = _f*_c1 + _i*_u;
		float _ch = lstmpeephole_activ_CH(_c);
		//
		float dch = dy[(mega_t*T+0+_t)*_lstm_VARS + depart_ch(X,Y) + _y];
		
		atomicAdd(&dy[(mega_t*T+0+_t)*_lstm_VARS + depart_c (X,Y) + _y], dch*d_lstmpeephole_activ_CH(_c,_ch));

		float dc = dy[(mega_t*T+0+_t)*_lstm_VARS + depart_c (X,Y) + _y];

		float d_f  = dc*_c1;
		float d_c1 = dc*_f;

		float d_i  = dc*_u;
		float d_u  = dc*_i;

		               atomicAdd(&dy[( mega_t   *T+0+_t)*_lstm_VARS + depart_f(X,Y) + _y], d_f);
		if (mega_t!=0) atomicAdd(&dy[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + _y], d_c1);
		               atomicAdd(&dy[( mega_t   *T+0+_t)*_lstm_VARS + depart_i(X,Y) + _y], d_i);
		               atomicAdd(&dy[( mega_t   *T+0+_t)*_lstm_VARS + depart_u(X,Y) + _y], d_u);
	}
}

static void d_lstm_peephole_o(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l,
	float * dy,
	float * dx,
	float * dp)
{
	float * x0=x; uint X0_vars=X_vars;         uint X0=X; uint depart_x0=DEPART_x;      float * dx0=dx; uint x0_depart__t= mega_t   *T;
	float * x1=y; uint X1_vars=lstm_VARS(X,Y); uint X1=Y; uint depart_x1=depart_h(X,Y); float * dx1=dy; uint x1_depart__t=(mega_t-1)*T;
	float * x2=y; uint X2_vars=lstm_VARS(X,Y); uint X2=Y; uint depart_x2=depart_c(X,Y); float * dx2=dy; uint x2_depart__t= mega_t   *T;
	//
	uint Y__vars=lstm_VARS (X,Y); uint depart__y=depart_o  (X,Y); uint y__depart__t=mega_t*T;
	uint L__vars=lstm_LOCDS(X,Y); uint depart__l=depart_dsO(X,Y);
	//
	float * a0=p; uint depart_a0=depart_poids_o(X,Y);						 float * da0=dp;
	float * a1=p; uint depart_a1=depart_poids_o(X,Y)+Ox(X,Y);				 float * da1=dp;
	float * a2=p; uint depart_a2=depart_poids_o(X,Y)+Ox(X,Y)+Oh(X,Y);		 float * da2=dp;
	float * b =p; uint depart__b=depart_poids_o(X,Y)+Ox(X,Y)+Oh(X,Y)+Oc(X,Y);float * db =dp;
	//
	uint activation = cuda_math_LOGISTIC;
	//
	if (mega_t == 0) {
		d_nvidia_F_AX_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, dx0, x0_depart__t,
			x2, X2_vars, X2, depart_x2, dx2, x2_depart__t,
			//
			y, Y__vars, Y, depart__y, dy, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0, da0,
			a2, depart_a2, da2,
			b , depart__b, db,
			//
			activation);
	} else {
		d_nvidia_F_AX_AX_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, dx0, x0_depart__t,
			x1, X1_vars, X1, depart_x1, dx1, x1_depart__t,
			x2, X2_vars, X2, depart_x2, dx2, x2_depart__t,
			//
			y, Y__vars, Y, depart__y, dy, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0, da0,
			a1, depart_a1, da1,
			a2, depart_a2, da2,
			b , depart__b, db,
			//
			activation);
	}
}

static __global__ void d_kerd_lstm1d_peephole_naive___partie_h(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l,
	float * dy,
	float * dx,
	float * dp)
{
	//	--- Partie h ---
	//h = o * ch

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float _o  = y[(mega_t*T+0+_t)*_lstm_VARS + depart_o (X,Y) + _y];
		float _ch = y[(mega_t*T+0+_t)*_lstm_VARS + depart_ch(X,Y) + _y];
		//
		//y[(mega_t*T+0+_t)*_lstm_VARS + depart_h(X,Y) + _y] = _o * _ch;
		//
		float _dy = dy[(mega_t*T+0+_t)*_lstm_VARS + depart_h(X,Y) + _y];// = _o * _ch;
		//
		atomicAdd(&dy[(mega_t*T+0+_t)*_lstm_VARS + depart_o (X,Y) + _y], _ch*_dy);
		//dy[(mega_t*T+0+_t)*_lstm_VARS + depart_o (X,Y) + _y] /*=*/+= _ch*_dy;
		atomicAdd(&dy[(mega_t*T+0+_t)*_lstm_VARS + depart_ch(X,Y) + _y], _o*_dy);
		//dy[(mega_t*T+0+_t)*_lstm_VARS + depart_ch(X,Y) + _y] /*=*/+= _o*_dy;
	}
}

void d_nvidia_lstm1d_peephole_shared_16_2(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l,
	float * dy,
	float * dx,
	float * dp)
{
	//h = o * ch
	d_kerd_lstm1d_peephole_naive___partie_h<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();


	//	o = logistique(sO = Ox@x + Oh@h + Oc@c     + Ob)
	d_lstm_peephole_o(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();


	//c = f*c[-1] + i*u
	//ch = tanh(c)
	d_kerd_lstm1d_peephole_naive___partie_cch<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();


	//f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
	//i = logistique(sI = Ix@x + Ih@h + Ic@c[-1] + Ib)
	//u =       tanh(sU = Ux@x + Uh@h +          + Ub)
	d_lstm_peephole_f(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l,
		dy,
		dx,
		dp); //mega_t==0 => df=0
	d_lstm_peephole_i(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l,
		dy,
		dx,
		dp);
	d_lstm_peephole_u(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
}