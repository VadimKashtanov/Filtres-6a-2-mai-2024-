#include "hip/hip_runtime.h"
#include "lstm1d_peephole.cuh"

#define BLOQUE_T 16
#define BLOQUE_Y 16

/*
//	--- Partie fiu ---
f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
i = logistique(sI = Ix@x + Ih@h + Ic@c[-1] + Ib)
u =       tanh(sU = Ux@x + Uh@h +          + Ub)
//	--- Partie cch ---
c = f*c[-1] + i*u
ch = tanh(c)
//	--- Partie o ---
o = logistique(sO = Ox@x + Oh@h + Oc@c    + Ob)
//	--- Partie h ---
h = o * ch
*/

static void lstm_peephole_f(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l)
{
	float * x0=x; uint X0_vars=X_vars;         uint X0=X; uint depart_x0=DEPART_x;      uint x0_depart__t=mega_t*T;
	float * x1=y; uint X1_vars=lstm_VARS(X,Y); uint X1=Y; uint depart_x1=depart_h(X,Y); uint x1_depart__t=(mega_t-1)*T;
	float * x2=y; uint X2_vars=lstm_VARS(X,Y); uint X2=Y; uint depart_x2=depart_c(X,Y); uint x2_depart__t=(mega_t-1)*T;
	//
	uint Y__vars=lstm_VARS (X,Y); uint depart__y=depart_f  (X,Y); uint y__depart__t=mega_t*T;
	uint L__vars=lstm_LOCDS(X,Y); uint depart__l=depart_dsF(X,Y);
	//
	float * a0=p; uint depart_a0=depart_poids_f(X,Y);
	float * a1=p; uint depart_a1=depart_poids_f(X,Y)+Fx(X,Y);
	float * a2=p; uint depart_a2=depart_poids_f(X,Y)+Fx(X,Y)+Fh(X,Y);
	float * b =p; uint depart__b=depart_poids_f(X,Y)+Fx(X,Y)+Fh(X,Y)+Fc(X,Y);
	//
	uint activation = cuda_math_LOGISTIC;
	//
	if (mega_t == 0) {
		//f = logistique(sF = Fx@x + Fh@h +          + Fb)
		nvidia_F_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, x0_depart__t,
			//
			y, Y__vars, Y, depart__y, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0,
			b , depart__b,
			//
			activation);
	} else {
		//f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
		nvidia_F_AX_AX_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, x0_depart__t,
			x1, X1_vars, X1, depart_x1, x1_depart__t,
			x2, X2_vars, X2, depart_x2, x2_depart__t,
			//
			y, Y__vars, Y, depart__y, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0,
			a1, depart_a1,
			a2, depart_a2,
			b , depart__b,
			//
			activation);
	}
}

static void lstm_peephole_i(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l)
{
	float * x0=x; uint X0_vars=X_vars;         uint X0=X; uint depart_x0=DEPART_x;      uint x0_depart__t=mega_t*T;
	float * x1=y; uint X1_vars=lstm_VARS(X,Y); uint X1=Y; uint depart_x1=depart_h(X,Y); uint x1_depart__t=(mega_t-1)*T;
	float * x2=y; uint X2_vars=lstm_VARS(X,Y); uint X2=Y; uint depart_x2=depart_c(X,Y); uint x2_depart__t=(mega_t-1)*T;
	//
	uint Y__vars=lstm_VARS (X,Y); uint depart__y=depart_i  (X,Y); uint y__depart__t=mega_t*T;
	uint L__vars=lstm_LOCDS(X,Y); uint depart__l=depart_dsI(X,Y);
	//
	float * a0=p; uint depart_a0=depart_poids_i(X,Y);
	float * a1=p; uint depart_a1=depart_poids_i(X,Y)+Ix(X,Y);
	float * a2=p; uint depart_a2=depart_poids_i(X,Y)+Ix(X,Y)+Ih(X,Y);
	float * b =p; uint depart__b=depart_poids_i(X,Y)+Ix(X,Y)+Ih(X,Y)+Ic(X,Y);
	//
	uint activation = cuda_math_LOGISTIC;
	//
	if (mega_t == 0) {
		//f = logistique(sF = Fx@x + Fh@h +          + Fb)
		nvidia_F_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, x0_depart__t,
			//
			y, Y__vars, Y, depart__y, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0,
			b , depart__b,
			//
			activation);
	} else {
		//f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
		nvidia_F_AX_AX_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, x0_depart__t,
			x1, X1_vars, X1, depart_x1, x1_depart__t,
			x2, X2_vars, X2, depart_x2, x2_depart__t,
			//
			y, Y__vars, Y, depart__y, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0,
			a1, depart_a1,
			a2, depart_a2,
			b , depart__b,
			//
			activation);
	}
}

static void lstm_peephole_u(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l)
{
	float * x0=x; uint X0_vars=X_vars;         uint X0=X; uint depart_x0=DEPART_x;      uint x0_depart__t=mega_t*T;
	float * x1=y; uint X1_vars=lstm_VARS(X,Y); uint X1=Y; uint depart_x1=depart_h(X,Y); uint x1_depart__t=(mega_t-1)*T;
	//
	uint Y__vars=lstm_VARS (X,Y); uint depart__y=depart_u  (X,Y); uint y__depart__t=mega_t*T;
	uint L__vars=lstm_LOCDS(X,Y); uint depart__l=depart_dsU(X,Y);
	//
	float * a0=p; uint depart_a0=depart_poids_u(X,Y);
	float * a1=p; uint depart_a1=depart_poids_u(X,Y)+Ux(X,Y);
	float * b =p; uint depart__b=depart_poids_u(X,Y)+Ux(X,Y)+Uh(X,Y);
	//
	uint activation = cuda_math_TANH;
	//
	if (mega_t == 0) {
		//f = logistique(sF = Fx@x + Fh@h +          + Fb)
		nvidia_F_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, x0_depart__t,
			//
			y, Y__vars, Y, depart__y, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0,
			b , depart__b,
			//
			activation);
	} else {
		//f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
		nvidia_F_AX_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, x0_depart__t,
			x1, X1_vars, X1, depart_x1, x1_depart__t,
			//
			y, Y__vars, Y, depart__y, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0,
			a1, depart_a1,
			b , depart__b,
			//
			activation);
	}
}

static __global__ void kerd_lstm1d_peephole_naive___partie_cch(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l)
{
	//	--- Partie c ---
	//c = f*c[-1] + i*u

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float _f  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_f(X,Y) + _y];
		float _c1 = (mega_t==0 ? 0.0 : y[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + _y]);
		float _i  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_i(X,Y) + _y];
		float _u  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_u(X,Y) + _y];
		//
		float _c = _f*_c1 + _i*_u;
		//
		y[(mega_t*T+_t)*_lstm_VARS + depart_c (X,Y) + _y] = _c;
		y[(mega_t*T+_t)*_lstm_VARS + depart_ch(X,Y) + _y] = lstmpeephole_activ_CH(_c);
	}
}

static void lstm_peephole_o(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l)
{
	float * x0=x; uint X0_vars=X_vars;         uint X0=X; uint depart_x0=DEPART_x;      uint x0_depart__t=mega_t*T;
	float * x1=y; uint X1_vars=lstm_VARS(X,Y); uint X1=Y; uint depart_x1=depart_h(X,Y); uint x1_depart__t=(mega_t-1)*T;
	float * x2=y; uint X2_vars=lstm_VARS(X,Y); uint X2=Y; uint depart_x2=depart_c(X,Y); uint x2_depart__t=mega_t*T;
	//
	uint Y__vars=lstm_VARS (X,Y); uint depart__y=depart_o  (X,Y); uint y__depart__t=mega_t*T;
	uint L__vars=lstm_LOCDS(X,Y); uint depart__l=depart_dsO(X,Y);
	//
	float * a0=p; uint depart_a0=depart_poids_o(X,Y);
	float * a1=p; uint depart_a1=depart_poids_o(X,Y)+Ox(X,Y);
	float * a2=p; uint depart_a2=depart_poids_o(X,Y)+Ox(X,Y)+Oh(X,Y);
	float * b =p; uint depart__b=depart_poids_o(X,Y)+Ox(X,Y)+Oh(X,Y)+Oc(X,Y);
	//
	uint activation = cuda_math_LOGISTIC;
	//
	if (mega_t == 0) {
		nvidia_F_AX_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, x0_depart__t,
			x2, X2_vars, X2, depart_x2, x2_depart__t,
			//
			y, Y__vars, Y, depart__y, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0,
			a2, depart_a2,
			b , depart__b,
			//
			activation);
	} else {
		nvidia_F_AX_AX_AX__shared_16(
			T,			//KERD(T)
			//
			x0, X0_vars, X0, depart_x0, x0_depart__t,
			x1, X1_vars, X1, depart_x1, x1_depart__t,
			x2, X2_vars, X2, depart_x2, x2_depart__t,
			//
			y, Y__vars, Y, depart__y, y__depart__t,
			l, L__vars,    depart__l,
			//
			a0, depart_a0,
			a1, depart_a1,
			a2, depart_a2,
			b , depart__b,
			//
			activation);
	}
}

static __global__ void kerd_lstm1d_peephole_naive___partie_h(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l)
{
	//	--- Partie h ---
	//h = o * ch

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float _o  = y[(mega_t*T+_t)*_lstm_VARS + depart_o (X,Y) + _y];
		float _ch = y[(mega_t*T+_t)*_lstm_VARS + depart_ch(X,Y) + _y];
		//
		y[(mega_t*T+0+_t)*_lstm_VARS + depart_h(X,Y) + _y] = _o * _ch;
	}
}

void nvidia_lstm1d_peephole_shared_16_2(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * l)
{
	//f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
	//i = logistique(sI = Ix@x + Ih@h + Ic@c[-1] + Ib)
	//u =       tanh(sU = Ux@x + Uh@h +          + Ub)
	lstm_peephole_f(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l); //mega_t==0 => df=0
	lstm_peephole_i(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l);
	lstm_peephole_u(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l);
	ATTENDRE_CUDA();


	//c = f*c[-1] + i*u
	//ch = tanh(c)
	kerd_lstm1d_peephole_naive___partie_cch<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l);
	ATTENDRE_CUDA();


	//	o = logistique(sO = Ox@x + Oh@h + Oc@c     + Ob)
	lstm_peephole_o(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l);
	ATTENDRE_CUDA();


	//h = o * ch
	kerd_lstm1d_peephole_naive___partie_h<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		l);
	ATTENDRE_CUDA();
}