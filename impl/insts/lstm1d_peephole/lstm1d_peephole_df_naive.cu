#include "hip/hip_runtime.h"
#include "lstm1d_peephole.cuh"

#define BLOQUE_T 16//32
#define BLOQUE_Y 16//32


static __global__ void d__kerd_lstm1d_peephole_naive___partie_h(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	//	--- Partie h ---
	//h = o * ch

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float _o  = y[(mega_t*T+0+_t)*_lstm_VARS + depart_o (X,Y) + _y];
		float _ch = y[(mega_t*T+0+_t)*_lstm_VARS + depart_ch(X,Y) + _y];
		//
		//y[(mega_t*T+0+_t)*_lstm_VARS + depart_h(X,Y) + _y] = _o * _ch;
		//
		float _dy = dy[(mega_t*T+0+_t)*_lstm_VARS + depart_h(X,Y) + _y];// = _o * _ch;
		//
		atomicAdd(&dy[(mega_t*T+0+_t)*_lstm_VARS + depart_o (X,Y) + _y], _ch*_dy);
		//dy[(mega_t*T+0+_t)*_lstm_VARS + depart_o (X,Y) + _y] /*=*/+= _ch*_dy;
		atomicAdd(&dy[(mega_t*T+0+_t)*_lstm_VARS + depart_ch(X,Y) + _y], _o*_dy);
		//dy[(mega_t*T+0+_t)*_lstm_VARS + depart_ch(X,Y) + _y] /*=*/+= _o*_dy;
	}
}

static __global__ void d__kerd_lstm1d_peephole_naive___partie_o(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	//	--- Partie och ---
	//o = logistique(sO = Ox@x + Oh@h + Oc@c + Ob)

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float _dy  =   dy[(mega_t*T+0+_t)*_lstm_VARS  + depart_o(X,Y)   + _y];
		float l_so = locd[(mega_t*T+0+_t)*_lstm_LOCDS + depart_dsO(X,Y) + _y];
		float ds   = _dy * l_so;

		//so += p[depart_poids_o(X,Y) + Ox(X,Y) + Oh(X,Y) + Oc(X,Y) + _y];
		atomicAdd(&dp[depart_poids_o(X,Y) + Ox(X,Y) + Oh(X,Y) + Oc(X,Y) + _y], ds);

		FOR(0, i_y, Y) {
			float _c = (mega_t==0 ? 0.0 : y[(mega_t    *T+0+_t)*_lstm_VARS + depart_c(X,Y) + i_y]);
			float _h = (mega_t==0 ? 0.0 : y[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_h(X,Y) + i_y]);
			//
			//so += _h * p[depart_poids_o(X,Y) + Ox(X,Y) + _y*Y + i_y];
			float d_h = p[depart_poids_o(X,Y) + Ox(X,Y) + _y*Y + i_y] * ds;
			atomicAdd(&dp[depart_poids_o(X,Y) + Ox(X,Y) + _y*Y + i_y], ds * _h);
			if (mega_t!=0) atomicAdd(&dy[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_h(X,Y) + i_y], d_h);
			//
			//so += _c * p[depart_poids_o(X,Y) + Ox(X,Y) + Oh(X,Y) + _y*Y + i_y];
			float d_c = p[depart_poids_o(X,Y) + Ox(X,Y) + Oh(X,Y) + _y*Y + i_y] * ds;
			atomicAdd(&dp[depart_poids_o(X,Y) + Ox(X,Y) + Oh(X,Y) + _y*Y + i_y], _c * ds);
			atomicAdd(&dy[(mega_t*T+0+_t)*_lstm_VARS + depart_c(X,Y) + i_y], d_c);
		}

		FOR(0, i_x, X) {	//x
			float _x = x[(mega_t*T+0+_t)*X_vars + DEPART_x + i_x];
			//so += _x * p[depart_poids_o(X,Y) + _y*X + i_x];
			float d_x = p[depart_poids_o(X,Y) + _y*X + i_x] * ds;
			atomicAdd(&dp[depart_poids_o(X,Y) + _y*X + i_x], _x*ds);
			atomicAdd(&dx[(mega_t*T+0+_t)*X_vars + DEPART_x + i_x], d_x);
		};
	}
}

static __global__ void d__kerd_lstm1d_peephole_naive___partie_cch(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	//	--- Partie c ---
	//c = f*c[-1] + i*u
	//ch = tanh(c)

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float _f  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_f(X,Y) + _y];
		float _c1 = (mega_t==0 ? 0.0 : y[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + _y]);
		float _i  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_i(X,Y) + _y];
		float _u  =                    y[( mega_t   *T+0+_t)*_lstm_VARS + depart_u(X,Y) + _y];
		//
		float _c = _f*_c1 + _i*_u;
		float _ch = lstmpeephole_activ_CH(_c);
		//
		float dch = dy[(mega_t*T+0+_t)*_lstm_VARS + depart_ch(X,Y) + _y];
		
		atomicAdd(&dy[(mega_t*T+0+_t)*_lstm_VARS + depart_c (X,Y) + _y], dch*d_lstmpeephole_activ_CH(_c,_ch));

		float dc = dy[(mega_t*T+0+_t)*_lstm_VARS + depart_c (X,Y) + _y];

		float d_f  = dc*_c1;
		float d_c1 = dc*_f;

		float d_i  = dc*_u;
		float d_u  = dc*_i;

		               atomicAdd(&dy[( mega_t   *T+0+_t)*_lstm_VARS + depart_f(X,Y) + _y], d_f);
		if (mega_t!=0) atomicAdd(&dy[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + _y], d_c1);
		               atomicAdd(&dy[( mega_t   *T+0+_t)*_lstm_VARS + depart_i(X,Y) + _y], d_i);
		               atomicAdd(&dy[( mega_t   *T+0+_t)*_lstm_VARS + depart_u(X,Y) + _y], d_u);
	}
}

static __global__ void d__kerd_lstm1d_peephole_naive___partie_fiu(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	//	--- Partie fiu ---
	//f = logistique(sF = Fx@x + Fh@h + Fc@c[-1] + Fb)
	//i = logistique(sI = Ix@x + Ih@h + Ic@c[-1] + Ib)
	//u =       tanh(sU = Ux@x + Uh@h +          + Ub)

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		uint _lstm_VARS  = lstm_VARS (X,Y);
		uint _lstm_LOCDS = lstm_LOCDS(X,Y);
		//
		float lf = locd[(mega_t*T+0+_t)*_lstm_LOCDS + depart_dsF(X,Y) + _y];
		float li = locd[(mega_t*T+0+_t)*_lstm_LOCDS + depart_dsI(X,Y) + _y];
		float lu = locd[(mega_t*T+0+_t)*_lstm_LOCDS + depart_dsU(X,Y) + _y];
		//
		float df = dy[(mega_t*T+0+_t)*_lstm_VARS + depart_f(X,Y) + _y];
		float di = dy[(mega_t*T+0+_t)*_lstm_VARS + depart_i(X,Y) + _y];
		float du = dy[(mega_t*T+0+_t)*_lstm_VARS + depart_u(X,Y) + _y];
		//
		float dsf = lf * df;
		float dsi = li * di;
		float dsu = lu * du;
		//
		atomicAdd(&dp[depart_poids_f(X,Y) + Fx(X,Y) + Fh(X,Y) + Fc(X,Y) + _y], dsf);
		atomicAdd(&dp[depart_poids_i(X,Y) + Ix(X,Y) + Ih(X,Y) + Ic(X,Y) + _y], dsi);
		atomicAdd(&dp[depart_poids_u(X,Y) + Ux(X,Y) + Uh(X,Y) +         + _y], dsu);
		//
		FOR(0, i_y, Y) {	//h & c
			float _c = (mega_t==0 ? 0.0 : y[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + i_y]);
			float _h = (mega_t==0 ? 0.0 : y[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_h(X,Y) + i_y]);

			//	-------------------------------------------------
			
			//sf += _h * p[depart_poids_f(X,Y) + Fx(X,Y) + _y*Y + i_y];
			float dsf_h = dsf * p[depart_poids_f(X,Y) + Fx(X,Y) + _y*Y + i_y];
			atomicAdd(&dp[depart_poids_f(X,Y) + Fx(X,Y) + _y*Y + i_y], dsf * _h);
			if (mega_t!=0) atomicAdd(&dy[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_h(X,Y) + i_y], dsf_h);

			//si += _h * p[depart_poids_i(X,Y) + Ix(X,Y) + _y*Y + i_y];
			float dsi_h = dsi * p[depart_poids_i(X,Y) + Ix(X,Y) + _y*Y + i_y];
			atomicAdd(&dp[depart_poids_i(X,Y) + Ix(X,Y) + _y*Y + i_y], dsi * _h);
			if (mega_t!=0) atomicAdd(&dy[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_h(X,Y) + i_y], dsi_h);
			
			//su += _h * p[depart_poids_u(X,Y) + Ux(X,Y) + _y*Y + i_y];
			float dsu_h = dsu * p[depart_poids_u(X,Y) + Ux(X,Y) + _y*Y + i_y];
			atomicAdd(&dp[depart_poids_u(X,Y) + Ux(X,Y) + _y*Y + i_y], dsu * _h);
			if (mega_t!=0) atomicAdd(&dy[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_h(X,Y) + i_y], dsu_h);
			
			//	-------------------------------------------------

			//sf += _c * p[depart_poids_f(X,Y) + Fx(X,Y) + Fh(X,Y) + _y*Y + i_y];
			float dsf_c = dsf * p[depart_poids_f(X,Y) + Fx(X,Y) + Fh(X,Y) + _y*Y + i_y];
			atomicAdd(&dp[depart_poids_f(X,Y) + Fx(X,Y) + Fh(X,Y) + _y*Y + i_y], dsf * _c);
			if (mega_t!=0) atomicAdd(&dy[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + i_y], dsf_c);
			
			//si += _c * p[depart_poids_i(X,Y) + Ix(X,Y) + Ih(X,Y) + _y*Y + i_y];
			float dsi_c = dsi * p[depart_poids_i(X,Y) + Ix(X,Y) + Ih(X,Y) + _y*Y + i_y];
			atomicAdd(&dp[depart_poids_i(X,Y) + Ix(X,Y) + Ih(X,Y) + _y*Y + i_y], dsi * _c);
			if (mega_t!=0) atomicAdd(&dy[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + i_y], dsi_c);
			
			//su += _c * p[depart_poids_u(X,Y) + Ux(X,Y) + Uh(X,Y) + _y*Y + i_y];
			//float dsu_c = dsu * p[depart_poids_u(X,Y) + Ux(X,Y) + Uh(X,Y) + _y*Y + i_y];
			//atomicAdd(&dp[depart_poids_u(X,Y) + Ux(X,Y) + Uh(X,Y) + _y*Y + i_y], dsu * _c);
			//if (mega_t!=0) atomicAdd(&dy[((mega_t-1)*T+0+_t)*_lstm_VARS + depart_c(X,Y) + i_y], dsu_c);
		};

		FOR(0, i_x, X) {	//x
			float _x = x[(mega_t*T+0+_t)*X_vars + DEPART_x + i_x];
			float _dx = 0;
			//sf += _x * p[depart_poids_f(X,Y) + _y*X + i_x];
			_dx += dsf * p[depart_poids_f(X,Y) + _y*X + i_x];
			atomicAdd(&dp[depart_poids_f(X,Y) + _y*X + i_x], _x * dsf);

			//si += _x * p[depart_poids_i(X,Y) + _y*X + i_x];
			_dx += dsi * p[depart_poids_i(X,Y) + _y*X + i_x];
			atomicAdd(&dp[depart_poids_i(X,Y) + _y*X + i_x], _x * dsi);

			//su += _x * p[depart_poids_u(X,Y) + _y*X + i_x];
			_dx += dsu * p[depart_poids_u(X,Y) + _y*X + i_x];
			atomicAdd(&dp[depart_poids_u(X,Y) + _y*X + i_x], _x * dsu);

			//dx
			atomicAdd(&dx[(mega_t*T+0+_t)*X_vars + DEPART_x + i_x], _dx);
		};
	}
};

void d_nvidia_lstm1d_peephole_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	//	--- DERIVE Partie h ---
	d__kerd_lstm1d_peephole_naive___partie_h<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();

	//	--- DERIVE Partie och ---
	d__kerd_lstm1d_peephole_naive___partie_o<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();

	//	--- DERIVE Partie c ---
	d__kerd_lstm1d_peephole_naive___partie_cch<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();

	//	--- DERIVE Partie fiu ---
	d__kerd_lstm1d_peephole_naive___partie_fiu<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
};