#include "hip/hip_runtime.h"
#include "filtres_prixs.cuh"

#define BLOQUE_T  4//8
#define BLOQUE_B  4//8
#define BLOQUE_FB 8//16//32//8

#include "../../../impl_tmpl/tmpl_etc.cu"

static __device__ float filtre_device(float * x, float * dif_x, float * f, float * locd) {
	float s = 0, d = 0;
	float f_nouveau = f[0];
	s += powf(1 + fabs(x[0] - f_nouveau), 0.5/*(0.5+0/N*0.5)*/);
	float f_avant   = f_nouveau;
	FOR(1, i, N) {
		f_nouveau = f[i];
		float Ps = 0.5;//(0.5+i/N*0.5);
		float Pd = 1.5;//(1.0+i/N*1.0);
		//s += powf(1 + fabs(   x[i]  -       f_nouveau    ), 0.5);
		//d += powf(1 + fabs(dif_x[i] - (f_nouveau-f_avant)), 2.0);
		s += powf(1 + fabs(   x[i]  -       f_nouveau    ), Ps);
		d += powf(1 + fabs(dif_x[i] - (f_nouveau-f_avant)), Pd);
		f_avant   = f_nouveau;
	};

	s = s/(float)N-1;
	d = d/(float)(N-1)-1;
	
	float y = expf(-s*s -d*d);

	locd[0] = -2*s*y*fp_d_normalisation(y);//-2*2*s*y;
	locd[1] = -2*d*y*fp_d_normalisation(y);//-2*2*d*y;

	//return 2*y-1;
	return fp_normalisation(y);
	//return 2*filtres_f_info(y)-1;
};

static __global__ void kerd_filtre_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint t0, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _b = threadIdx.y + blockIdx.y * blockDim.y;
	uint _f = threadIdx.z + blockIdx.z * blockDim.z;

	if (_t < T && _b < bloques && _f < f_par_bloque) {
		uint depart_plus_t = t_MODE(
			_t_MODE, GRAINE,
			t0, t0+T*MEGA_T,
			_t, mega_t,
			T, MEGA_T
		);
		//
		y[(mega_t*T + 0+_t)*(bloques*f_par_bloque) + _b*f_par_bloque + _f] = filtre_device(
			x     + _b*PRIXS*N_FLTR + depart_plus_t*N_FLTR,
			dif_x + _b*PRIXS*N_FLTR + depart_plus_t*N_FLTR,
			f     + _b*f_par_bloque*N     + _f*N,
			locd  + (mega_t*T + 0+_t)*bloques*f_par_bloque*2 + _b*f_par_bloque*2 + _f*2
		);
	}
};

void nvidia_filtres_prixs___naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint t0, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd)
{
	kerd_filtre_naive<<<dim3(KERD(T, BLOQUE_T), KERD(bloques, BLOQUE_B), KERD(f_par_bloque, BLOQUE_FB)), dim3(BLOQUE_T, BLOQUE_B, BLOQUE_FB)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		t0, T,
		bloques, f_par_bloque,
		x, dif_x,
		f,
		y,
		locd);
	ATTENDRE_CUDA();
}

__device__ static void d_nvidia_filtre(float * x, float * dif_x, float * f, float * locd, float * dy, float * df) {
	/*float ds = locd[0] * dy[0] / 8;
	float dd = locd[1] * dy[0] / 7;
	//
	FOR(1, i, N)
	{
		//s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		atomicAdd(&df[i], ds * 1 / (2*sqrtf(1 + fabs(x[i] - f[i]))) * (-1) * cuda_signe(x[i] - f[i]));
		//d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
		atomicAdd(&df[ i ], dd * 2 * (1 + fabs(dif_x[i] - (f[i]-f[i-1]))) * cuda_signe(dif_x[i] - (f[i]-f[i-1])) * (-1));
		atomicAdd(&df[i-1], dd * 2 * (1 + fabs(dif_x[i] - (f[i]-f[i-1]))) * cuda_signe(dif_x[i] - (f[i]-f[i-1])) * (+1));
	}
	atomicAdd(&df[0], ds * 1 / (2*sqrtf(1 + fabs(x[0] - f[0]))) * (-1) * cuda_signe(x[0] - f[0]));*/

	float ds = locd[0] * dy[0] / 8;
	float dd = locd[1] * dy[0] / 7;
	//
	FOR(1, i, N)
	{
		float Ps = 0.5;//(0.5+i/N*0.5);
		float Pd = 1.5;//(1.0+i/N*1.0);
		
		//s += powf(1 + fabs(   x[i]  -       f_nouveau    ), (0.5+i/N*0.5));
		atomicAdd(&df[i], ds * Ps * powf(1 + fabs(x[i] - f[i]), Ps-1) * (-1) * cuda_signe(x[i] - f[i]));
		//d += powf(1 + fabs(dif_x[i] - (f_nouveau-f_avant)), (1.0+i/N*1.0));
		atomicAdd(&df[ i ], dd * Pd * powf(1 + fabs(dif_x[i] - (f[i]-f[i-1])), Pd-1) * cuda_signe(dif_x[i] - (f[i]-f[i-1])) * (-1));
		atomicAdd(&df[i-1], dd * Pd * powf(1 + fabs(dif_x[i] - (f[i]-f[i-1])), Pd-1) * cuda_signe(dif_x[i] - (f[i]-f[i-1])) * (+1));

	}
	float Ps = 0.5;//(0.5+0/N*0.5);
	//df[0] += ds * 1 / (2*sqrtf(1 + fabs(x[0] - f[0]))) * (-1) * signe(x[0] - f[0]);
	atomicAdd(&df[0], ds * Ps * powf(1 + fabs(x[0] - f[0]), Ps-1) * (-1) * cuda_signe(x[0] - f[0]));

};

__global__ static void  d_nvidia_kerd_filtre_naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint t0, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd,
	float * dy,
	float * df)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _b = threadIdx.y + blockIdx.y * blockDim.y;
	uint _f = threadIdx.z + blockIdx.z * blockDim.z;

	if (_t < T && _b < bloques && _f < f_par_bloque) {
		uint depart_plus_t = t_MODE(
			_t_MODE, GRAINE,
			t0, t0+T*MEGA_T,
			_t, mega_t,
			T, MEGA_T
		);
		//
		d_nvidia_filtre(
				x + _b*PRIXS*N_FLTR + depart_plus_t*N_FLTR,
			dif_x + _b*PRIXS*N_FLTR + depart_plus_t*N_FLTR,
			f     + _b*f_par_bloque*N     + _f*N,
			locd  + (mega_t*T+0+_t)*(bloques*f_par_bloque*2) + _b*(f_par_bloque*2) + _f*2,
			dy    + (mega_t*T+0+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f,
			df    + _b*f_par_bloque*N     + _f*N
		);
	}
};

void d_nvidia_filtres_prixs___naive(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint t0, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd,
	float * dy,
	float * df)
{
	d_nvidia_kerd_filtre_naive<<<dim3(KERD(T, BLOQUE_T), KERD(bloques, BLOQUE_B), KERD(f_par_bloque, BLOQUE_FB)), dim3(BLOQUE_T, BLOQUE_B, BLOQUE_FB)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		t0, T,
		bloques, f_par_bloque,
		x, dif_x,
		f,
		y,
		locd,
		dy,
		df);
	ATTENDRE_CUDA();
}