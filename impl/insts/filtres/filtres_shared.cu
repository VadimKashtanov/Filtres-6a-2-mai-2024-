#include "hip/hip_runtime.h"
#include "filtres_prixs.cuh"

#define BLOQUE_T  8

#define _repete_T 8

#include "../../../impl_tmpl/tmpl_etc.cu"

static __global__ void kerd_filtre_shared(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint t0, uint T,
	uint bloques,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd)
{
	uint depart_bloque_t = blockIdx.y * BLOQUE_T * _repete_T;
	uint depart_thread_t = depart_bloque_t + threadIdx.y * _repete_T;

	uint _b = blockIdx.x;
	uint _f = blockIdx.z;	//(ligne dans bloque)

	uint LIGNE  = _b;
	uint BLOQUE = _b; 

	uint thx = threadIdx.x;
	uint thy__t = threadIdx.y;

	//if (_t < T)
	__shared__ float __f__[N];
	//
	if (thy__t==0) __f__[thx]  = f[BLOQUE*F_PAR_BLOQUES*N + _f*N + thx];
	__syncthreads();
	//
	float fi, fi1;
	fi = __f__[thx];
	if (thx != 0)
		fi1 = __f__[thx-1];
	//
	__shared__ float __ret[BLOQUE_T][2];	//s, d
	__shared__ float __y  [BLOQUE_T];
	//
	float xi, dif_xi;
	//
	uint _t;
	FOR(0, plus_t, _repete_T) {
		//
		_t = depart_thread_t + plus_t;
		//uint cuda_depart_plus_t = t_MODE_GENERALE_cuda(_t_MODE, GRAINE, depart, DEPART, FIN, _t, MEGA_T)+mega_t;
		uint depart_plus_t = t_MODE(
			_t_MODE, GRAINE,
			t0, t0+T*MEGA_T,
			_t, mega_t,
			T, MEGA_T
		);
		//
		if (thx < 2) {
			__ret[thy__t][thx] = 0;
		}
		__syncthreads();
		//
		xi = x[LIGNE*PRIXS*N_FLTR + depart_plus_t*N_FLTR + thx];
		//
		if (thx != 0) {
			float Pd = 1.5;//(1.0+thx/N*1.0);
			dif_xi = dif_x[LIGNE*PRIXS*N_FLTR + depart_plus_t*N_FLTR + thx];
			atomicAdd(&__ret[thy__t][1], powf((1 + fabs(dif_xi - (fi-fi1))), Pd));
		}
		float Ps = 0.5;//(0.5+thx/N*0.5);
		atomicAdd(&__ret[thy__t][0], powf(1 + fabs(xi - fi), Ps));
		__syncthreads();
		//
		if (thx < 2) {
			__ret[thy__t][thx] = __ret[thy__t][thx]/(float)(8-thx) - 1.0;
		}
		__syncthreads();
		//
		if (thx < 1) {
			__y[thy__t] = expf(-__ret[thy__t][0]*__ret[thy__t][0] -__ret[thy__t][1]*__ret[thy__t][1]);
		}
		__syncthreads();
		//
		if (thx < 2) {
			//locd[(0+_t)*BLOQUES*(F_PAR_BLOQUES*2) + BLOQUE*(F_PAR_BLOQUES*2) + _f*2 + thx] = -2*2*__ret[thy__t][thx]*__y[thy__t];
			//locd[(0+_t)*BLOQUES*(F_PAR_BLOQUES*2) + BLOQUE*(F_PAR_BLOQUES*2) + _f*2 + thx] = -2*2*__ret[thy__t][thx]*__y[thy__t];
			locd[(mega_t*T+0+_t)*BLOQUES*(F_PAR_BLOQUES*2) + BLOQUE*(F_PAR_BLOQUES*2) + _f*2 + thx] = -2*__ret[thy__t][thx]*__y[thy__t]*fp_d_normalisation(__y[thy__t]);

		}
		__syncthreads();
		//
		if (thx < 1) {
			//y[(0+_t)*BLOQUES*F_PAR_BLOQUES + BLOQUE*F_PAR_BLOQUES + _f] = 2*__y[thy__t] - 1;
			//y[(0+_t)*BLOQUES*F_PAR_BLOQUES + BLOQUE*F_PAR_BLOQUES + _f] = 2*__y[thy__t] - 1;
			y[(mega_t*T+0+_t)*BLOQUES*F_PAR_BLOQUES + BLOQUE*F_PAR_BLOQUES + _f] = fp_normalisation(__y[thy__t]);

		}
	};
};

void nvidia_filtres_prixs___shared(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint t0, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd)
{
	ASSERT(BLOQUE_T*_repete_T <= T);
	kerd_filtre_shared<<<dim3(bloques, KERD((DIV(T,_repete_T)), BLOQUE_T), F_PAR_BLOQUES), dim3(N, BLOQUE_T,1)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		t0, T,
		bloques,
		x, dif_x,
		f,
		y,
		locd);
	ATTENDRE_CUDA();
};

static __global__ void d_kerd_filtre_shared(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint t0, uint T,
	uint bloques,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd,
	float * dy,
	float * df)
{
	uint depart_bloque_t = blockIdx.y * BLOQUE_T * _repete_T;
	uint depart_thread_t = depart_bloque_t + threadIdx.y * _repete_T;

	uint _b = blockIdx.x;
	uint _f = blockIdx.z;	//(ligne dans bloque)

	uint LIGNE  = _b;
	uint BLOQUE = _b; 

	uint thx = threadIdx.x;
	uint thy__t = threadIdx.y;

	//if (_t < T)
	__shared__ float __f__[N];
	__shared__ float __df__[N];
	//
	if (thy__t==0) {
		__f__[thx]  = f[BLOQUE*F_PAR_BLOQUES*N + _f*N + thx];
		__df__[thx] = 0;
	}
	__syncthreads();
	//
	float fi, fi1;
	fi = __f__[thx];
	if (thx != 0)
		fi1 = __f__[thx-1];
	//
	__shared__ float __locd[BLOQUE_T][2];	//ds, dd
	__shared__ float __dy0[BLOQUE_T];
	//
	float xi, dif_xi;
	float tmp;
	//
	uint _t;
	FOR(0, plus_t, _repete_T) {
		_t = depart_thread_t + plus_t;
		//uint cuda_depart_plus_t = t_MODE_GENERALE_cuda(_t_MODE, GRAINE, depart, DEPART, FIN, _t, MEGA_T)+mega_t;
		uint depart_plus_t = t_MODE(
			_t_MODE, GRAINE,
			t0, t0+T*MEGA_T,
			_t, mega_t,
			T, MEGA_T
		);
		//
		if (thx < 1) {
			__dy0[thy__t] = dy[(mega_t*T+0+_t)*BLOQUES*F_PAR_BLOQUES + BLOQUE*F_PAR_BLOQUES + _f];
		}
		__syncthreads();
		//
		if (thx < 2) {
			__locd[thy__t][thx] = locd[(mega_t*T+0+_t)*BLOQUES*(F_PAR_BLOQUES*2) + BLOQUE*(F_PAR_BLOQUES*2) + _f*2 + thx] * __dy0[thy__t]/ (float)(8 - thx);
		}
		__syncthreads();
		//
		xi = x[LIGNE*PRIXS*N_FLTR + depart_plus_t*N_FLTR + thx];
		//
		if (thx != 0) {
			dif_xi = dif_x[LIGNE*PRIXS*N_FLTR + depart_plus_t*N_FLTR + thx];
			//atomicAdd(&__ret[thy__t][1], powf((1 + fabs(dif_xi - (fi-fi1))), 2));
			float Pd = 1.5;//(1.0+thx/N*1.0);
			tmp = (Pd) * powf(1 + fabs(dif_xi - (fi-fi1)), Pd-1) * cuda_signe(dif_xi - (fi-fi1));
			atomicAdd(&__df__[ thx ], __locd[thy__t][1] * tmp * (-1));
			atomicAdd(&__df__[thx-1], __locd[thy__t][1] * tmp * (+1));
		}
		//atomicAdd(&__ret[thy__t][0], sqrtf(1 + fabs(xi - fi)));
		float Ps = 0.5;//(0.5+thx/N*0.5);
		atomicAdd(&__df__[thx], __locd[thy__t][0] * (Ps) * powf(1 + fabs(xi - fi), Ps-1) * (-1) * cuda_signe(xi - fi));
		__syncthreads();
	};
	__syncthreads();
	if (thy__t == 0) {
		atomicAdd(&df[BLOQUE*F_PAR_BLOQUES*N + _f*N + thx], __df__[thx]);
	}
};

void d_nvidia_filtres_prixs___shared(
	uint mega_t,
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint t0, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd,
	float * dy,
	float * df)
{
	ASSERT(BLOQUE_T*_repete_T <= T);
	d_kerd_filtre_shared<<<dim3(bloques, KERD((DIV(T,_repete_T)), BLOQUE_T), F_PAR_BLOQUES), dim3(N, BLOQUE_T,1)>>>(
		mega_t,
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		t0, T,
		bloques,
		x, dif_x,
		f,
		y,
		locd,
		dy,
		df);
	ATTENDRE_CUDA();
}