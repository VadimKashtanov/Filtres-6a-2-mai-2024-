#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

char * nom_sources[SOURCES] = {
	" prixs    BTC",
	"  haut    BTC",
	"  bas     BTC",
	" median   BTC",
	"volumes   BTC",
	"volumes A BTC",
	"volumes U BTC",
	//
	" prixs    ETH",
	"  haut    ETH",
	"  bas     ETH",
	" median   ETH"
	"volumes   ETH",
	"volumes A ETH",
	"volumes U ETH",
};

//	Sources
float     prixs_btc[PRIXS] = {};
float      high_btc[PRIXS] = {};
float       low_btc[PRIXS] = {};
float    median_btc[PRIXS] = {};
float   volumes_btc[PRIXS] = {};
float volumes_A_btc[PRIXS] = {};
float volumes_U_btc[PRIXS] = {};
//
float     prixs_eth[PRIXS] = {};
float      high_eth[PRIXS] = {};
float       low_eth[PRIXS] = {};
float    median_eth[PRIXS] = {};
float   volumes_eth[PRIXS] = {};
float volumes_A_eth[PRIXS] = {};
float volumes_U_eth[PRIXS] = {};


float *     prixs_btc__d = 0x0;
float *      high_btc__d = 0x0;
float *       low_btc__d = 0x0;
float *    median_btc__d = 0x0;
float *   volumes_btc__d = 0x0;
float * volumes_A_btc__d = 0x0;
float * volumes_U_btc__d = 0x0;
//
float *     prixs_eth__d = 0x0;
float *      high_eth__d = 0x0;
float *       low_eth__d = 0x0;
float *    median_eth__d = 0x0;
float *   volumes_eth__d = 0x0;
float * volumes_A_eth__d = 0x0;
float * volumes_U_eth__d = 0x0;


float * sources[SOURCES] = {
	prixs_btc, high_btc, low_btc, median_btc, volumes_btc, volumes_A_btc, volumes_U_btc,
	prixs_eth, high_eth, low_eth, median_eth, volumes_eth, volumes_A_eth, volumes_U_eth
};

float * sources__d[SOURCES] = {
	prixs_btc__d, high_btc__d, low_btc__d, median_btc__d, volumes_btc__d, volumes_A_btc__d, volumes_U_btc__d,
	prixs_eth__d, high_eth__d, low_eth__d, median_eth__d, volumes_eth__d, volumes_A_eth__d, volumes_U_eth__d
};

static void charger_une_source(float * ou, char * fichier) {
	FILE * fp = fopen(fichier, "rb");
	ASSERT(fp != 0);
	uint __PRIXS;
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(ou, sizeof(float), PRIXS, fp);
	fclose(fp);
};

void charger_les_prixs() {
	//	--- BTC ---
	charger_une_source(prixs_btc,     "prixs/BTCUSDT/prixs.bin"    );
	charger_une_source(high_btc,      "prixs/BTCUSDT/high.bin"     );
	charger_une_source(low_btc,       "prixs/BTCUSDT/median.bin"   );
	charger_une_source(median_btc,    "prixs/BTCUSDT/median.bin"   );
	charger_une_source(volumes_btc,   "prixs/BTCUSDT/volumes.bin"  );
	charger_une_source(volumes_A_btc, "prixs/BTCUSDT/volumes_A.bin");
	charger_une_source(volumes_U_btc, "prixs/BTCUSDT/volumes_U.bin");
	
	//	--- ETH ---
	charger_une_source(prixs_eth,     "prixs/ETHUSDT/prixs.bin"    );
	charger_une_source(high_eth,      "prixs/ETHUSDT/high.bin"     );
	charger_une_source(low_eth,       "prixs/ETHUSDT/median.bin"   );
	charger_une_source(median_eth,    "prixs/ETHUSDT/median.bin"   );
	charger_une_source(volumes_eth,   "prixs/ETHUSDT/volumes.bin"  );
	charger_une_source(volumes_A_eth, "prixs/ETHUSDT/volumes_A.bin");
	charger_une_source(volumes_U_eth, "prixs/ETHUSDT/volumes_U.bin");
};

//	===========================================================

void ema_int_calc_ema(ema_int_t * ema_int) {
	//			-- Parametres --
	uint K = ema_int->K_ema;
	float _K = 1.0 / ((float)K);
	//	EMA
	ema_int->ema[0] = sources[ema_int->source][0];
	FOR(1, i, PRIXS) {
		ema_int->ema[i] = ema_int->ema[i-1] * (1.0 - _K) + sources[ema_int->source][i]*_K;
	}
};

//	===========================================================

uint nature_multiple_interv[NATURES] = {
	0,
	0,
	0,
	14,
	14
};

nature_f fonctions_nature[NATURES] = {
	nature0__direct,
	nature1__macd,
	nature2__chiffre,
	nature3__awesome,
	nature4__pourcent_r,
	nature5__rsi,
};

uint NATURE_PARAMS[NATURES] = {
	0,
	1,
	1,
	1,
	2,
	2
};

uint min_param[NATURES][MAX_PARAMS] = {
	{0,0,0,0},
	{1,0,0,0},
	{1,0,0,0},
	{1,0,0,0},
	{1,1,0,0},
	{1,1,0,0}
};

uint max_param[NATURES][MAX_PARAMS] = {
	{0,                0,       0,        0      }, 
	{MAX_COEF_MACD,    0,       0,        0      },
	{MAX_CHIFFRE,      0,       0,        0      },
	{MAX_COEF_AWESOME, 0,       0,        0      },
	{MAX_INTERVALLE,   MAX_EMA, 0,        0      },
	{MAX_INTERVALLE,   MAX_EMA, 0,        0      } 
};

char * nom_natures[NATURES] {
	"directe",
	"  macd ",
	"chiffre",
	"awesome",
	"  %R   ",
	"  RSI  "
};

ema_int_t * cree_ligne(uint source, uint nature, uint K_ema, uint intervalle, uint params[MAX_PARAMS]) {
	ema_int_t * ret = alloc<ema_int_t>(1);
	//
	ret->source = source;
	ret->nature = nature;
	ret->K_ema  = K_ema;
	ret->intervalle = intervalle;
	//
	ASSERT(intervalle <= MAX_INTERVALLE);
	ASSERT(K_ema      <= MAX_EMA);
	//
	memcpy(ret->params, params, sizeof(uint) * MAX_PARAMS);
	//
	ema_int_calc_ema(ret);
	fonctions_nature[nature](ret);
	//
	return ret;
};

void liberer_ligne(ema_int_t * ema_int) {

};

void charger_vram_nvidia() {
	prixs_btc__d     = cpu_vers_gpu<float>(prixs_btc,     PRIXS);
	high_btc__d      = cpu_vers_gpu<float>(high_btc,      PRIXS);
	low_btc__d       = cpu_vers_gpu<float>(low_btc,       PRIXS);
	median_btc__d    = cpu_vers_gpu<float>(median_btc,    PRIXS);
	volumes_btc__d   = cpu_vers_gpu<float>(volumes_btc,   PRIXS);
	volumes_A_btc__d = cpu_vers_gpu<float>(volumes_A_btc, PRIXS);
	volumes_U_btc__d = cpu_vers_gpu<float>(volumes_U_btc, PRIXS);
	//
	prixs_eth__d     = cpu_vers_gpu<float>(prixs_eth,     PRIXS);
	high_eth__d      = cpu_vers_gpu<float>(high_eth,      PRIXS);
	low_eth__d       = cpu_vers_gpu<float>(low_eth,       PRIXS);
	median_eth__d    = cpu_vers_gpu<float>(median_eth,    PRIXS);
	volumes_eth__d   = cpu_vers_gpu<float>(volumes_eth,   PRIXS);
	volumes_A_eth__d = cpu_vers_gpu<float>(volumes_A_eth, PRIXS);
	volumes_U_eth__d = cpu_vers_gpu<float>(volumes_U_eth, PRIXS);
};

void     liberer_cudamalloc() {
	CONTROLE_CUDA(hipFree(  prixs_btc__d));
	CONTROLE_CUDA(hipFree(   high_btc__d));
	CONTROLE_CUDA(hipFree(    low_btc__d));
	CONTROLE_CUDA(hipFree( median_btc__d));
	CONTROLE_CUDA(hipFree(volumes_btc__d));
	CONTROLE_CUDA(hipFree(volumes_A_btc__d));
	CONTROLE_CUDA(hipFree(volumes_U_btc__d));
	//
	CONTROLE_CUDA(hipFree(  prixs_eth__d));
	CONTROLE_CUDA(hipFree(   high_eth__d));
	CONTROLE_CUDA(hipFree(    low_eth__d));
	CONTROLE_CUDA(hipFree( median_eth__d));
	CONTROLE_CUDA(hipFree(volumes_eth__d));
	CONTROLE_CUDA(hipFree(volumes_A_eth__d));
	CONTROLE_CUDA(hipFree(volumes_U_eth__d));
};

void charger_tout() {
	//	Assertions
	FOR(0, i, NATURES) ASSERT(nature_multiple_interv[i] <= MAX_MULTPLE_INTERV_NATURES);
	//
	printf("charger_les_prixs : ");    MESURER(charger_les_prixs());
	printf("charger_vram_nvidia : ");  MESURER(charger_vram_nvidia());
};

void liberer_tout() {
	titre("Liberer tout");
	liberer_cudamalloc();
};

ema_int_t * lire_ema_int(FILE * fp) {
	uint source, nature, K_ema, intervalle;
	uint params[MAX_PARAMS];
	FREAD(&source,     sizeof(uint), 1, fp);
	FREAD(&nature,     sizeof(uint), 1, fp);
	FREAD(&K_ema,      sizeof(uint), 1, fp);
	FREAD(&intervalle, sizeof(uint), 1, fp);
	//
	FREAD(&params,     sizeof(uint), MAX_PARAMS, fp);
	//
	return cree_ligne(source, nature, K_ema, intervalle, params);
};

void      ecrire_ema_int(ema_int_t * ema_int, FILE * fp) {
	FWRITE(&ema_int->source,     sizeof(uint), 1, fp);
	FWRITE(&ema_int->nature,     sizeof(uint), 1, fp);
	FWRITE(&ema_int->K_ema,      sizeof(uint), 1, fp);
	FWRITE(&ema_int->intervalle, sizeof(uint), 1, fp);
	//
	FWRITE(&ema_int->params,     sizeof(uint), MAX_PARAMS, fp);
};

char * nom_type_de_norme[3] = {
	"NORME_CLASSIQUE",
	"NORME_THEORIQUE",
	"NORME_RELATIVE "
};