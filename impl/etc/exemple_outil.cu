#include "hip/hip_runtime.h"
#include "marchee.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

static void ecrire_courbe(char * fichier, float * courbe, uint L) {
	FILE * fp = fopen(fichier, "wb");
	//
	//FWRITE(&L, sizeof(uint), 1, fp);
	FWRITE(courbe, sizeof(float), L, fp);
	//
	fclose(fp);
};

void visualiser_ema_int(
	uint source,
	uint nature,
	uint K_ema, uint intervalle,
	uint params[MAX_PARAMS])
{
	ema_int_t * ema_int = cree_ligne(
		source, nature, K_ema, intervalle, params);
	//
	float un_filtre[PRIXS] = {0};
	FOR(0, i, PRIXS) un_filtre[i] = 0;
	uint depart = PRIXS - 1 - intervalle*N_FLTR;
	float s = 0;
	float r[N_FLTR];
	FOR(0, i, N_FLTR) {
		s += rnd()-.5;
		r[i] = s;
	}
	float max=r[0], min=r[0];
	FOR(0, i, N_FLTR) {
		if (r[i] > max) max = r[i];
		if (r[i] < min) min = r[i];
	}
	FOR(0, i, N_FLTR) r[i] = (r[i]-min)/(max-min);
	//
	//
	/*max=0; min=0;
	FOR(0, i, PRIXS) {
		if (ema_int->brute[i] > max) max = ema_int->brute[i];
		if (ema_int->brute[i] < min) min = ema_int->brute[i];
	};*/
	max = ema_int->brute[PRIXS-1]; 
	min = ema_int->brute[PRIXS-1-N_FLTR*intervalle];
	FOR(0, i, N_FLTR-1) {
		FOR(0, j, intervalle) {
			float val = r[i] + (r[i+1]-r[i]) * (j / (float)intervalle);
			un_filtre[depart + i*intervalle + j] = val*(max-min) + min;
		}
	}
	//un_filtre[depart + N_FLTR*intervalle] = r[N_FLTR-1]*(max-min) + min;
	//
	//
	ecrire_courbe("tmp_source.float", sources[source], PRIXS);
	ecrire_courbe("tmp_ema.float",       ema_int->ema, PRIXS);
	ecrire_courbe("tmp_brute.float",   ema_int->brute, PRIXS);
	ecrire_courbe("tmp_filtre.float",       un_filtre, PRIXS);
	//
	char cmd[100];
	snprintf(cmd, 100, "python3 afficher_exemple_outil.py %i &", PRIXS);
	//
	SYSTEM(cmd);
	//
	liberer_ligne(ema_int);
}