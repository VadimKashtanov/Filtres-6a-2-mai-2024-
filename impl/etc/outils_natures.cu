#include "hip/hip_runtime.h"
#include "marchee.cuh"

void _outil_ema(float * y, float * x, uint K) {
	float _K = 1 / ((float)K);
	y[0] = x[0];
	FOR(1, t, PRIXS) {
		y[t] = y[t-1]*(1 - _K) + x[t] * _K;
	}
};

void _outil_macd(float * y, float * x, float coef) {
	ASSERT(coef > 0.0);
	float ema12[PRIXS], ema26[PRIXS], ema9[PRIXS], __macd[PRIXS];
	_outil_ema(ema12, x, 12*coef);
	_outil_ema(ema26, x, 26*coef);
	FOR(0, i, PRIXS) __macd[i] = ema12[i] - ema26[i];
	_outil_ema(ema9, __macd, 12*coef);
	FOR(0, i, PRIXS) y[i] = __macd[i] - ema9[i];
};

void _outil_chiffre(float * y, float * x, float chiffre) {
	FOR(0, t, PRIXS) {
		//y[t] = 2*(chiffre-MIN2(fabs(x[t]-chiffre*roundf((x[t]+0)/chiffre)), fabs(x[t]-chiffre*roundf((x[t]+chiffre)/chiffre))))/chiffre-1;
		float haut =  ceil(x[t]/chiffre)*chiffre;
		float bas  = floor(x[t]/chiffre)*chiffre;
		if (haut == bas) haut += chiffre;
		y[t] = 2*((x[t]-bas)/(haut-bas)-0.5);
	}
};

void _outil_awesome(float * y, float * x, float coef) {
	ASSERT(coef > 0.0);
	float ema5[PRIXS], ema30[PRIXS];
	_outil_ema( ema5, x,  5*coef);
	_outil_ema(ema30, x, 34*coef);
	FOR(0, i, PRIXS) y[i] = ema5[i] - ema30[i];
};

void _outil_pourcent_r(float * y, float * x, uint interv, uint ema_post_r) {
	ASSERT(interv > 0);
	uint n = 14*interv;
	FOR(0, i, n) y[i] = 0.0;
	FOR(n, i, PRIXS) {
		float max=x[i], min=x[i];
		FOR(0, j, n) {
			if (x[i-j] > max) max = x[i-j];
			if (x[i-j] < min) min = x[i-j];
		}
		y[i] = (max - x[i])/(max - min)  * (-1);	//%r a pour nature d'etre negatif
	};
	//
	//	Petit Ema post %R, pour lisser (ou pas) la courbe chaotique
	float _K = 1 / ((float)ema_post_r);
	FOR(1, t, PRIXS) {
		y[t] = y[t-1]*(1 - _K) + y[t] * _K;
	}
};

void _outil_rsi(float * y, float * x, uint interv) {
	ASSERT(interv > 0);
	uint n = 14*interv;
	FOR(0, i, n+1) y[i] = 0.0;
	//
	float changements[PRIXS];
	FOR(1, i, PRIXS) changements[i] = x[i] - x[i-1];
	//
	//#pragma omp parallel
	//#pragma omp for
	FOR(n+1, t, PRIXS) {
		float  gain_moy = 0;
		float perte_moy = 0;
		FOR(0, i, n) {
			if (changements[t-i] >= 0) gain_moy  += +(changements[t-i]);
			if (changements[t-i] <  0) perte_moy += -(changements[t-i]);
		}
		if (perte_moy != 0) {
			float rs = (gain_moy/n) / (perte_moy/n);
			y[t] = 1.0 - 1.0/(1+rs);
		} else {
			y[t] = 1.0;
		}
	}
};