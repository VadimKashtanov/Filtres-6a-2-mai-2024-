#include "hip/hip_runtime.h"
#include "cuda_math.cuh"

#define BLOQUE     16
#define BLOQUE_MAX BLOQUE//16

#include "./activations.cu"

//y = activation(a0@x0 + a1@x1 + a2@x2 + b)

static __global__ void kerd_f_ax_b__shared_16__t(
	uint T,			//KERD(T)
	//
	float * x0, uint X0_vars, uint X0, uint depart_x0, uint x0_depart__t,
	//
	float *  y, uint Y__vars, uint  Y, uint depart__y, uint y__depart__t,
	float *  l, uint L__vars,          uint depart__l,
	//
	float * a0, uint depart_a0,
	float * b , uint depart__b,
	//
	uint activation)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	//	+a0@x0
	FOR(0, d, X0/BLOQUE) {
		__partage__x[thy][thx] = x0[(x0_depart__t+_t)*X0_vars + depart_x0 + (d*BLOQUE + thx)];
		__partage__p[thy][thx] = a0[   depart_a0             + _y*X0     + (d*BLOQUE + thy)];
		__syncthreads();
#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

	//	+b
#define __partage__b __partage__x[0]
	if (thy == 0) __partage__b[thx] = b[depart__b + _y];
	__syncthreads();

	s = (s + __partage__b[thx]);
	float a = activation_f(activation, s);
	y[(y__depart__t+_t)*Y__vars + depart__y + _y] = a;
	l[(y__depart__t+_t)*L__vars + depart__l + _y] = activation_df(activation, s,a);
};

void nvidia_F_AX__shared_16(
	uint T,			//KERD(T)
	//
	float * x0, uint X0_vars, uint X0, uint depart_x0, uint x0_depart__t,
	//
	float *  y, uint Y__vars, uint  Y, uint depart__y, uint y__depart__t,
	float *  l, uint L__vars,          uint depart__l,
	//
	float * a0, uint depart_a0,
	float * b , uint depart__b,
	//
	uint activation)
{
	ASSERT(T%BLOQUE==0);
	ASSERT(X0%BLOQUE==0);
	ASSERT(Y%BLOQUE==0);
	//
	kerd_f_ax_b__shared_16__t<<<dim3(KERD(Y, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
		T,
		//
		x0, X0_vars, X0, depart_x0, x0_depart__t,
		//
		y, Y__vars, Y, depart__y, y__depart__t,
		l, L__vars,    depart__l,
		//
		a0, depart_a0,
		b , depart__b,
		//
		activation);
	//ATTENDRE_CUDA();
}

//	======================================================================================
//	======================================================================================
//	======================================================================================

static __global__ void d_kerd_f_ax_b__shared_16___dX( //pour a b c
	uint T,
	//
	float * x, uint X_vars, uint X, uint depart_x, float * dx,  uint x_depart__t,
	//
	float *  y, uint Y__vars, uint  Y, uint depart__y, float * dy, uint y__depart__t,
	float *  l, uint L__vars,          uint depart__l,
	//
	float * a, uint depart_a,	float * da,
	//
	uint activation)
{
	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _x = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, Y/BLOQUE) {
		float _l  = l[(y__depart__t+_t)*L__vars + depart__l + (d*BLOQUE+thx)];
		float _dy =   dy[(y__depart__t+_t)*Y__vars + depart__y + (d*BLOQUE+thx)];
		__partage__x[thy][thx] =  _l * _dy;
		__partage__p[thy][thx] = a[depart_a + (d*BLOQUE+thy)*X + _x];
		__syncthreads();
#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

	atomicAdd(&dx[(x_depart__t+_t)*X_vars + depart_x + _x], s);
};

static __global__ void d_kerd_f_ax_b__shared_16___dA(
	uint T,
	//
	float * x, uint X_vars, uint X, uint depart_x, float * dx, uint x_depart__t,
	//
	float *  y, uint Y__vars, uint  Y, uint depart__y, float * dy, uint y__depart__t,
	float *  l, uint L__vars,          uint depart__l,
	//
	float * a, uint depart_a,	float * da,
	//
	uint activation)
{
	//dp = x.T @ ((y-_y)*dtanh(x@p))

	__shared__ float __partage__x[BLOQUE_MAX][BLOQUE_MAX];
	__shared__ float __partage__p[BLOQUE_MAX][BLOQUE_MAX];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _x = thx + blockIdx.x * blockDim.x;
	uint _y = thy + blockIdx.y * blockDim.y;

	float s = 0;

	uint d = blockIdx.z;
	//FOR(0, d, T/BLOQUE) {
		float __l =  l[(y__depart__t+d*BLOQUE_MAX+thx)*L__vars + depart__l + _y];
		float _dy = dy[(y__depart__t+d*BLOQUE_MAX+thx)*Y__vars + depart__y + _y];
		__partage__x[thy][thx] = __l * _dy; 
		__partage__p[thy][thx] = x[(x_depart__t+(d*BLOQUE_MAX+thy))*X_vars + depart_x + _x];
		__syncthreads();

	#pragma unroll
		FOR(0, i, BLOQUE_MAX) {
			s += __partage__x[thy][i] * __partage__p[i][thx];
		}
		__syncthreads();
	//};

	atomicAdd(&da[depart_a + _y*X + _x], s);
};

static __global__ void d_kerd_f_ax_b__shared_16___db(
	uint T,
	//
	float *  y, uint Y__vars, uint  Y, uint depart__y, float * dy, uint y__depart__t,
	float *  l, uint L__vars,          uint depart__l,
	//
	float * b , uint depart__b, float * db,
	//
	uint activation)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		float _l  =  l[(y__depart__t+_t)*L__vars + depart__l + _y];
		float _dy = dy[(y__depart__t+_t)*Y__vars + depart__y + _y];
		atomicAdd(&db[depart__b + _y], _l * _dy);
	}
};

void d_nvidia_F_AX__shared_16(
	uint T,
	//
	float * x0, uint X0_vars, uint X0, uint depart_x0, float * dx0, uint x0_depart__t,
	//
	float *  y, uint Y__vars, uint  Y, uint depart__y, float * dy, uint y__depart__t,
	float *  l, uint L__vars,          uint depart__l,
	//
	float * a0, uint depart_a0,	float * da0,
	float * b , uint depart__b, float * db,
	//
	uint activation)
{
	ASSERT(T%BLOQUE==0);
	ASSERT(X0%BLOQUE==0);
	ASSERT(Y%BLOQUE==0);
	
	//dx0 = (a0 @ (dy*dtanh(x@p)).T).T
	d_kerd_f_ax_b__shared_16___dX<<<dim3(KERD(X0, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
		T,
		//
		x0, X0_vars, X0, depart_x0, dx0, x0_depart__t,
		//
		y, Y__vars, Y, depart__y, dy, y__depart__t,
		l, L__vars,    depart__l,
		//
		a0, depart_a0, da0,
		//
		activation);

	// ==============================================================================

	d_kerd_f_ax_b__shared_16___dA<<<dim3(KERD(X0, BLOQUE_MAX), KERD(Y, BLOQUE_MAX), DIV(T,BLOQUE_MAX)), dim3(BLOQUE_MAX, BLOQUE_MAX,1)>>>(
		T,
		//
		x0, X0_vars, X0, depart_x0, dx0, x0_depart__t,
		//
		y, Y__vars, Y, depart__y, dy, y__depart__t,
		l, L__vars,    depart__l,
		//
		a0, depart_a0, da0,
		//
		activation);

	//	=============================================================================

	d_kerd_f_ax_b__shared_16___db<<<dim3(KERD(Y, BLOQUE_MAX), KERD(T,BLOQUE_MAX)), dim3(BLOQUE_MAX, BLOQUE_MAX)>>>(
		T,
		//
		y, Y__vars, Y, depart__y, dy, y__depart__t,
		l, L__vars,    depart__l,
		//
		b , depart__b, db,
		//
		activation);
}