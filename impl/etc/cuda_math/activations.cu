#include "hip/hip_runtime.h"
#include "cuda_math.cuh"

static __device__ float  activation_f(uint act, float x) {
	if (act == 0) return 1 / (1 + expf(-x));
	else if (act == 1) return tanh(x);
	else assert(0);
};

static __device__ float activation_df(uint act, float x, float a) {
	if (act == 0) return a*(1-a);
	else if (act == 1) return 1 - a*a;
	else assert(0);
	return 0;
};