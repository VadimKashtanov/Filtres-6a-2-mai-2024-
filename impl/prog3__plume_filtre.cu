#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

/*
Utilisation :
	./prog0__plume_filtre mdl.bin bloque f_dans_bloque
*/

int main(int argc, char ** argv) {
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");  charger_tout();
	//
	if (argc == 4) {
		Mdl_t * mdl = ouvrire_mdl(GRAND_T, argv[1]);
		//
		char cmd[1000];
		//
		uint bloque = atoi(argv[2]);
		uint      f = atoi(argv[3]);
		//
		if (N_FLTR == 8) {
			uint depart = bloque*F_PAR_BLOQUES*N + f*N;
			snprintf(cmd, 1000, "python3 -c \"import matplotlib.pyplot as plt;plt.plot([%f,%f,%f,%f,%f,%f,%f,%f]);plt.show()\"",
				mdl->p[0][depart + 0],
				mdl->p[0][depart + 1],
				mdl->p[0][depart + 2],
				mdl->p[0][depart + 3],
				mdl->p[0][depart + 4],
				mdl->p[0][depart + 5],
				mdl->p[0][depart + 6],
				mdl->p[0][depart + 7]
			);
		} else if (N_FLTR == 4) {
			uint depart = bloque*F_PAR_BLOQUES*N + f*N;
			snprintf(cmd, 1000, "python3 -c \"import matplotlib.pyplot as plt;plt.plot([%f,%f,%f,%f]);plt.show()\"",
				mdl->p[0][depart + 0],
				mdl->p[0][depart + 1],
				mdl->p[0][depart + 2],
				mdl->p[0][depart + 3]
			);
		} else {
			ERR("Pas de N_FLTR==%i", N_FLTR);
		}
		//
		printf("Type de norme = %s\n", nom_type_de_norme[mdl->bloque[bloque]->type_de_norme]);
		//
		SYSTEM(cmd);
		liberer_mdl(mdl);
		//
	} else {
		ERR("./prog0__plume_filtre mdl.bin bloque f_dans_bloque")
	}
	liberer_tout();
};