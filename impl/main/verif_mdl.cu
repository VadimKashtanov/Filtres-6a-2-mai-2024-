#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

//static float _pourcent_masque_nulle[C] = {0};
//static float _alpha[C] = {0.01};

__global__
static void kerd_p1e5(float * p, uint i, float _1E5) {
	p[i] += _1E5;
};

static void p1e5(Mdl_t * mdl, uint c, uint i, float _1E5, uint _MODE) {
	kerd_p1e5<<<1,1>>>(mdl->p__d[c], i, _1E5);
	ATTENDRE_CUDA();
};

static void __performance() {
	/*ASSERT(C == 11);
	titre("Performance");
	//
	uint Y[C] = {
		512,
		256,
		256,
		256,
		128,
		64,
		32,
		16,
		8,
		4,
		P
	};
	uint insts[C] = {
		FILTRES_PRIXS,
		LSTM1D,
		LSTM1D,
		LSTM1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D
	};
	uint lignes[BLOQUES] = {0};
	FOR(0, i, BLOQUES) lignes[i] = rand() % EMA_INTS;
	Mdl_t * mdl = cree_mdl(Y, insts, lignes);
	plumer_mdl(mdl);
	//
	uint plus_T = 16*16*25;
	//
	mdl_plume_grad(mdl, DEPART, DEPART+plus_T);
	//
	printf("TEMPS MODEL = ");
	MESURER(mdl_aller_retour(mdl, DEPART, DEPART+plus_T, 3));
	//
	liberer_mdl(mdl);*/
};

static void __verif_mdl_1e5() {
	/*ASSERT(SCORE_Y_COEF_BRUIT == 0.0);
	ASSERT(C == 3);
	titre("Comparer MODEL 1e-5");
	//
	uint Y[C];
	uint insts[C];
	//
	uint st[C][2] = {
		{64, FILTRES_PRIXS},
		{64, DOT1D_TANH},//LSTM1D_PEEPHOLE},
		//{64, LSTM1D_PEEPHOLE},
		{1,  DOT1D_TANH},
	};
	FOR(0, i, C) {
		    Y[i] = st[i][0];
		insts[i] = st[i][1];
	}
	ema_int_t * bloque[BLOQUES] = {
	//			  Source,      Nature,  K_ema, Intervalle,    {params}
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 1, 4, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 1, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 1, 4, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 1, 1, cree_DIRECTE())
	};
	ASSERT(F_PAR_BLOQUES == 8);
	ASSERT(BLOQUES == 8);
	//
	//
	uint plus_T = 8*8*1;
	//
	//
	Mdl_t * mdl = cree_mdl(plus_T, Y, insts, bloque);
	plumer_mdl(mdl);
	//
	//comportement(mdl, DEPART, DEPART+plus_T);
	//
	srand(0);
	uint MODE_t_MODE  = t_CONTINUE;//t_PSEUDO_ALEA;//t_CONTINUE;
	uint grain_t_MODE = rand() % 10000;
	//
#define MODE 1 //MODE_NAIF, MODE_MAXIMALE
	//
	printf("aller_retour T=%i : ", plus_T);
	MESURER(mdl_aller_retour(mdl, DEPART, DEPART+plus_T, MODE, MODE_t_MODE, grain_t_MODE, 1));
	//exit(0);
	//
	//	1e-5
	//
	INIT_CHRONO(temps_execution);
	DEPART_CHRONO(temps_execution);
	//
	mdl_zero_gpu(mdl);
	float _f = mdl_score(mdl, DEPART, DEPART+plus_T, MODE, MODE_t_MODE, grain_t_MODE);
	float _1E5 = 1e-2f;
	FOR(0, c, C) {
		printf("###############################################################\n");
		printf("                       C = %2.i (%s)    \n", c, nom_inst[mdl->insts[c]]);
		printf("#######################vvvvvvvvvvvvvv##########################\n");
		//
		float * dp = gpu_vers_cpu<float>(mdl->dp__d[c], mdl->inst_POIDS[c]);
		//
		FOR(0, i, mdl->inst_POIDS[c]) {
			p1e5(mdl, c, i, +_1E5, MODE);
			float grad_1e5 = (mdl_score(mdl, DEPART, DEPART+plus_T, MODE, MODE_t_MODE, grain_t_MODE) - _f)/_1E5;
			p1e5(mdl, c, i, -_1E5, MODE);
			//
			float a=grad_1e5, b=dp[i];
			plume_separateur_p(mdl, c, i);
			printf("%i| ", i);
			PLUME_CMP(a, b);
			if (b != 0) printf(" (x%f) ", a/b);
			printf("\n");
		}
		//
		free(dp);
	};
	printf("  1e5 === df(x)  \n");

	printf("Temps total = %f\n", VALEUR_CHRONO(temps_execution));

	//
	liberer_mdl(mdl);*/
};

void verif_mdl_1e5() {
	__performance();
	__verif_mdl_1e5();
};