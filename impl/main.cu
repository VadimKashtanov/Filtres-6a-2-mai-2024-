#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

static uint mise_a_jour_fichier(char * fichier, float score)
{
	FILE * fp = fopen(fichier, "rb");
	//
	uint meilleur = true;
	//
	if (fp == 0) {
		fp = fopen(fichier, "wb");
		ecrire_un<uint>(fp, 1);
		ecrire_un<float>(fp, score);
		fclose(fp);
	} else {
		uint I = lire_un<uint>(fp);
		float * scores = lire<float>(fp, I);
		fclose(fp);
		//
		meilleur = score PLUS_PETIT_QUE scores[I-1];
		//
		fp = fopen(fichier, "wb");
		ecrire_un<uint>(fp, I+1);
		ecrire<float>(fp, scores, I);
		ecrire_un<float>(fp, score);
		fclose(fp);
		//
		free(scores);
	};
	//
	return meilleur;
};

static void visualiser() {
	uint source     = SRC_PRIXS_BTC;
	uint nature     = POURCENT_R;
	uint K_ema      = 64;
	uint intervalle = 128;
	uint * params   = cree_POURCENT_R(128, 2);
	visualiser_ema_int(
		source,
		nature,
		K_ema, intervalle,
		params);
};

static void plume_pred(Mdl_t * mdl, uint t0, uint t1) {
	Stats_t * stats = statistiques(mdl, t0, t1);
	//
	printf("PRED GENERALE = %f%% | LES GAINS^2 = %f%% | LES GAINS^4 = %f%% | LES GAINS^8 = %f%%\n",
		100*stats->pred,
		100*stats->les_gains__2,
		100*stats->les_gains__4,
		100*stats->les_gains__8
	);
	//
	mise_a_jour_fichier("mdl", stats->score);
	//
	free(stats);
};

static void enregistrer_par_validation(Mdl_t * mdl) {
	Stats_t * stats = statistiques(mdl, DEPART_VALIDATION, FIN_VALIDATION);
	//
	printf("Validation = %f%% (score=%f) | Les gains^2 = %f%% | Les gains^4 = %f%% | Les gains^8 = %f%%\n", 
		100*stats->pred,
		    stats->score,
		100*stats->les_gains__2,
		100*stats->les_gains__4,
		100*stats->les_gains__8
	);
	//
	if (mise_a_jour_fichier("mdl_validation", stats->score) == 1) {
		ecrire_mdl(mdl, "mdl_validation.bin");
	};
	//
	free(stats);
	printf("Fin validation\n");
};

float * pourcent_masque_nulle = de_a(0.0, 0.0, C);
float * pourcent_masque_opti_nulle = de_a(0.0, 0.0, C);

float * pourcent_masque = de_a(0.10, 0.10, C);			//	Des poids nulls
float * pourcent_masque_opti = de_a(0.30, 0.00, C);		//	Des poids qui ne s'optimiseront pas

float * alpha = de_a(1e-5, 1e-5, C);//de_a(1e-5, 1e-5, C);

uint * optimiser_tous_les = UNIFORME_C(1);

PAS_OPTIMISER()
int main(int argc, char ** argv) {
	pourcent_masque[C-1] = 0.0;
	alpha[0] = 1e-2;

	/*	
	/!\ /!\ /!\	/!\ /!\ /!\	/!\ /!\ /!\	/!\ /!\ /!\	/!\ /!\ /!\
	- Cloner edt.uvsq.fr et faire son emploie du temps
ou page statique sur telephone
	- Entree bruit
	- sans ou avec regression normalisation
	- !!! Pouvoire verifier gain usd de validation sans internet et tester_mdl.py
		/!\ /!\ /!\	/!\ /!\ /!\	/!\ /!\ /!\	/!\ /!\ /!\	/!\ /!\ /!\
	*/

	//// - Chiffre "Par le haut", "Par le bas". Tres precis. 90%=0.5, 99%=0.8
	//// - Filtres fausses erreures x0.1

	//pourcent_masque_nulle[0] = 0.00;//0.20
	//pourcent_masque[0] = 0.01;

	//	----- Lien constants ------
	
	//	-- Init --
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");   charger_tout();

	//	-- Verification --
	titre("Verifier MDL");     verif_mdl_1e5();

	//===============
	titre("  Programme Generale  ");
	ecrire_structure_generale("structure_generale.bin");

	//visualiser();

	/*uint     Y[C];
	uint insts[C];
	//
	uint st[C][2] = {
	//	{4096, DOT1D},
	//
	//    Y  ,   inst
		{2048, FILTRES_PRIXS},
		{128,  DOT1D_TANH},
		//
		{256,  DOT1D_TANH_ELMAN},
		{256,  DOT1D_TANH_ELMAN},
		{256,  DOT1D_TANH_ELMAN},
		{256,  DOT1D_TANH_ELMAN},
		{256,  DOT1D_TANH_ELMAN},
		{256,  DOT1D_TANH_ELMAN},
		{256,  DOT1D_TANH_ELMAN},
		//
		{1,    DOT1D_TANH},
	};
	FOR(0, i, C) {
		    Y[i] = st[i][0];
		insts[i] = st[i][1];
	}
	//
	//	Assurances :
	ema_int_t * bloque[BLOQUES] = {
	//			    Source,      Nature,  K_ema, Intervalle,     {params}
	// -------
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 2, 1.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 2, 2, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 2, 4, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 4, 2.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 4, 4, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 8, 4.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 8, 16, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 16, 16, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 16, 32, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 32, 16.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 32, 32, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 32, 64, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 64, 32.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 64, 64, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 64, 128, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 128, 64.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 128, 128, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 128, 256, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 256, 128.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 256, 256, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, DIRECT, 512, 256.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 2, 1.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 2, 2, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 2, 4, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 4, 2.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 4, 4, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 8, 4.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 8, 16, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 16, 16, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 16, 32, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 32, 16.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 32, 32, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 32, 64, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 64, 32.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 64, 64, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 64, 128, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 128, 64.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 128, 128, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 128, 256, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 256, 128.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 256, 256, cree_DIRECTE()),
		cree_ligne(SRC_HIGH_BTC, DIRECT, 512, 256.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 2, 1.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 2, 2, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 2, 4, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 4, 2.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 4, 4, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 8, 4.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 8, 16, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 16, 16, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 16, 32, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 32, 16.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 32, 32, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 32, 64, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 64, 32.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 64, 64, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 64, 128, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 128, 64.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 128, 128, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 128, 256, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 256, 128.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 256, 256, cree_DIRECTE()),
		cree_ligne(SRC_LOW_BTC, DIRECT, 512, 256.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 2, 1.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 2, 2, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 2, 4, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 4, 2.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 4, 4, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 8, 4.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 8, 16, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 16, 16, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 16, 32, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 32, 16.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 32, 32, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 32, 64, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 64, 32.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 64, 64, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 64, 128, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 128, 64.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 128, 128, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 128, 256, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 256, 128.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 256, 256, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_BTC, DIRECT, 512, 256.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 2, 1.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 2, 2, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 2, 4, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 4, 2.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 4, 4, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 8, 4.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 8, 16, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 16, 16, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 16, 32, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 32, 16.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 32, 32, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 32, 64, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 64, 32.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 64, 64, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 64, 128, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 128, 64.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 128, 128, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 128, 256, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 256, 128.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 256, 256, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_A_BTC, DIRECT, 512, 256.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 2, 1.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 2, 2, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 2, 4, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 4, 2.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 4, 4, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 8, 4.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 8, 16, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 16, 16, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 16, 32, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 32, 16.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 32, 32, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 32, 64, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 64, 32.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 64, 64, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 64, 128, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 128, 64.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 128, 128, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 128, 256, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 256, 128.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 256, 256, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES_U_BTC, DIRECT, 512, 256.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS_BTC, MACD, 1, 1, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 1, 2, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 1, 2, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 1, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 1, 8, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 1, 8, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 1, 8, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 2, 1.0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 2, 2, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 2, 2, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 2, 4, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 2, 4, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 2, 4, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 2, 16, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 2, 16, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 2, 16, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 2, 16, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 2.0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 2.0, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 4, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 4, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 4, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 8, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 8, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 8, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 32, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 32, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 32, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 4, 32, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 4.0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 4.0, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 4.0, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 8, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 8, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 8, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 16, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 16, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 16, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 16, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 64, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 64, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 64, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 8, 64, cree_MACD(64)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 8.0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 8.0, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 8.0, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 8.0, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 16, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 16, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 16, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 16, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 32, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 32, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 32, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 32, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 128, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 128, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 128, cree_MACD(64)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 16, 128, cree_MACD(128)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 16.0, cree_MACD(2)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 16.0, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 16.0, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 16.0, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 32, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 32, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 32, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 32, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 64, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 64, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 64, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 64, cree_MACD(64)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 256, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 256, cree_MACD(64)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 256, cree_MACD(128)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 32, 256, cree_MACD(256)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 32.0, cree_MACD(4)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 32.0, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 32.0, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 32.0, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 64, cree_MACD(8)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 64, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 64, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 64, cree_MACD(64)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 128, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 128, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 128, cree_MACD(64)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 64, 128, cree_MACD(128)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 128.0, cree_MACD(16)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 128.0, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 128.0, cree_MACD(64)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 128.0, cree_MACD(128)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 256, cree_MACD(32)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 256, cree_MACD(64)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 256, cree_MACD(128)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 256, cree_MACD(256)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 256, cree_MACD(64)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 256, cree_MACD(128)),
		cree_ligne(SRC_PRIXS_BTC, MACD, 256, 256, cree_MACD(256)),
	};
	//
	system("rm mdl");system("rm mdl_validation");
	Mdl_t * mdl = cree_mdl(GRAND_T, Y, insts, bloque);*/

	Mdl_t * mdl = ouvrire_mdl(GRAND_T, "mdl.bin");

	//mdl_re_cree_poids(mdl);

	//uint c=5, nouveau_Y=64;
	//mdl_changer_couche_Y(mdl, c, nouveau_Y);

	enregistrer_les_lignes_brute(mdl, "lignes_brute.bin");

	plumer_mdl(mdl);

	//	================= Initialisation ==============
	uint t0 = DEPART;
	uint t1 = FIN;
	//
	plume_pred(mdl, t0, t1);
	//comportement(mdl, t0, t0+GRAND_T);
	//
	srand(time(NULL));
#define PERTURBATIONS 0
	//
	printf("Parametres : alpha[0]=%e GRAND_T=%i P_S=%f\n", alpha[0], GRAND_T, P_S);
	//
	uint MODE_t_MODE  = 0;
	uint grain_t_MODE = 0;
	//
	uint rep = 0;
	while ( 1 ) {
		//
		//MODE_t_MODE = t_CONTINUE;
		MODE_t_MODE = t_PSEUDO_ALEA;
		//
		grain_t_MODE = rand() % 10000;
		//
		//
		optimisation_mini_packet(
			mdl,
			t0, t1,
			alpha, 1.0,
			ADAM, 100,
			//
			pourcent_masque,
			//pourcent_masque_nulle,
			//
			//pourcent_masque_opti,
			pourcent_masque_opti_nulle,
			//
			PERTURBATIONS,
			optimiser_tous_les,
			MODE_t_MODE, grain_t_MODE);
		//
		mdl_poids_gpu_vers_cpu(mdl);
		//
		ecrire_mdl(mdl, "mdl.bin");
		//
		enregistrer_par_validation(mdl);
		//
		if (rep % 10 == 0) plume_pred(mdl, t0, t1);
		//
		printf("===================================================\n");
		printf("================= TERMINE %i ======================\n", rep++);
		printf("===================================================\n");
	}
	//
	mdl_gpu_vers_cpu(mdl);
	ecrire_mdl(mdl, "mdl.bin");
	liberer_mdl(mdl);

	//	-- Fin --
	liberer_tout();
};