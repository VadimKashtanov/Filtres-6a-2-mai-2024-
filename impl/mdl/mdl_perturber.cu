#include "hip/hip_runtime.h"
#include "mdl.cuh"

static uint couche_aleatoire(Mdl_t * mdl) {
	uint a = rand() % mdl->total_POIDS;
	FOR(0, i, C) {
		if (a < mdl->inst_POIDS[i]) {
			return i;
		} else {
			a -= mdl->inst_POIDS[i];
		}
	}
	return C-1;
}

//	===================================================

static void perturber_filtre(Mdl_t * mdl) {
	uint f = rand() % (BLOQUES*F_PAR_BLOQUES);
	//
	float r[N];
	r[0] = rnd();
	FOR(1, i, N) r[i] = r[i-1] + rnd()-.5;
	//
	float coef = 0.90;
	FOR(0, i, N) mdl->p[0][f*N + i] = mdl->p[0][f*N + i]*coef + (1-coef)*r[i];
};

void perturber_filtres(Mdl_t * mdl, uint L) {
	FOR(0, i, L) perturber_filtre(mdl);
};

//	===================================================

static void perturber_echanger   (Mdl_t * mdl, uint c) {
	if (mdl->insts[c] != FILTRES_PRIXS) {
		uint X=mdl->Y[c-1], Y=mdl->Y[c];
		uint p0 = (X+1)*(rand()%Y) + (rand()%X);
		uint p1 = (X+1)*(rand()%Y) + (rand()%X);
		float vp0 = mdl->p[c][p0], vp1 = mdl->p[c][p1];
		mdl->p[c][p0] = vp1;
		mdl->p[c][p1] = vp0;
	}
};

static void perturber_diviser  (Mdl_t * mdl, uint c) {
	if (mdl->insts[c] != FILTRES_PRIXS) {
		uint X=mdl->Y[c-1], Y=mdl->Y[c];
		mdl->p[c][(X+1)*(rand()%Y) + (rand()%X)] /= (float)(1+rand()%3);
	}
};

static void perturber_plus_rnd  (Mdl_t * mdl, uint c) {
	if (mdl->insts[c] != FILTRES_PRIXS) {
		uint X=mdl->Y[c-1], Y=mdl->Y[c];
		uint pos = (X+1)*(rand()%Y) + (rand()%X);
		mdl->p[c][pos] += 0.1 * fabs(mdl->p[c][pos]) * 2*(rnd()-0.5);
	}
};

//	======================================

void perturber(Mdl_t * mdl, uint L) {
	mdl_poids_gpu_vers_cpu(mdl);
	FOR(0, i, L) {
		uint c_alea = couche_aleatoire(mdl);
		if (c_alea != 0) {
			float ch = rnd();
			if (0.00 <= ch && ch <= 0.01) perturber_echanger(mdl, c_alea);	// 1%
			if (0.01 <= ch && ch <= 0.10) perturber_diviser (mdl, c_alea);	// 9%
			if (0.10 <= ch && ch <= 1.00) perturber_plus_rnd(mdl, c_alea);	//90%
		} else {
			perturber_filtre(mdl);
		}
	}
	mdl_poids_cpu_vers_gpu(mdl);
	//
	mdl_normer_les_filtres(mdl);
};