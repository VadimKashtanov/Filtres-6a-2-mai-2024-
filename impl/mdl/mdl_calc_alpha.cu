#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

float mdl_moy_dp(Mdl_t * mdl, uint c) {
	uint POIDS = mdl->inst_POIDS[c];
	//
	float * dp = gpu_vers_cpu<float>(mdl->dp__d[c],	POIDS);
	float moy = 0;
	FOR(0, i, POIDS) moy += fabs(dp[i]);
	free(dp);
	return moy / (float)POIDS;
};