#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "filtres_prixs.cuh"
#include "dot1d_tanh.cuh"
#include "lstm1d_peephole.cuh"
#include "dot1d_tanh_elman.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

mdl_inst_f cree_inst[INSTS] = {
	cree_filtres_prixs,
	cree_dot1d_tanh,
	cree_lstm1d_peephole,
	cree_dot1d_tanh_elman
};

mdl_f_f inst_f [INSTS] = {
	f_filtres_prixs,
	f_dot1d_tanh,
	f_lstm1d_peephole,
	f_dot1d_tanh_elman
};

mdl_f_f inst_df[INSTS] = {
	df_filtres_prixs,
	df_dot1d_tanh,
	df_lstm1d_peephole,
	df_dot1d_tanh_elman
};

char * nom_inst[INSTS] = {
	"filtres_prixs      ",
	"dot1d tanh(x)      ",
	"lstmd1d_peephole   ",
	"dot1d tanh(x) elman"
};

mdl_inst_f plume_inst[INSTS] = {
	plume_filtres_prixs,
	plume_dot1d_tanh,
	plume_lstm1d_peephole,
	plume_dot1d_tanh_elman
};

mdl_inst_f regulariser_inst[INSTS] = {
	regulariser_filtres_prixs,
	regulariser_dot1d_tanh,
	regulariser_lstm1d_peephole,
	regulariser_dot1d_tanh_elman
};

uint * UNIFORME_C(uint x) {
	uint * ret = alloc<uint>(C);
	FOR(0, i, C) ret[i] = x;
	return ret;
};

static void calculer_normalisee__et__dif_normalisee(Mdl_t * mdl) { 
	FOR(0, b, BLOQUES) {
		FOR(DEPART, t, FIN) {
			//	_max & _min pour ce filtre-8
			float _max, _min;
			
			if (mdl->type_de_norme[b] == NORME_CLASSIQUE) {
				_max = mdl->bloque[b]->brute[t - 0*mdl->bloque[b]->intervalle];
				_min = mdl->bloque[b]->brute[t - 0*mdl->bloque[b]->intervalle];
				FOR(1, i, N_FLTR) {
					float xi = mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle];
					if (_max < xi)
						_max = xi;
					if (_min > xi)
						_min = xi;
				}


			} else if (mdl->type_de_norme[b] == NORME_RELATIVE) {
				_max = fabs(mdl->bloque[b]->brute[t - 0*mdl->bloque[b]->intervalle]);
				FOR(1, i, N_FLTR) {
					float xi = fabs(mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle]);
					if (_max < xi)
						_max = xi;
				}
				_max = +_max;
				_min = -_max;

			} else if (mdl->type_de_norme[b] == NORME_THEORIQUE) {
				_min = /*1.0;//*/mdl->min_theorique[b];
				_max = /*0.0;//*/mdl->max_theorique[b];
				FOR(0, i, N_FLTR) {
					float xi = mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle];
					if (!(_min <= xi && xi <= _max))
						printf("%f  %f %f\n", xi, _min, _max);
					ASSERT(_min <= xi && xi <= _max);
				}

			} else {
				ERR("Norme ni classique, ni theorique, mdl->type_de_norme[b] == %i", mdl->type_de_norme[b]);
			};

			if (_min == _max) {
				FOR(0, i, N_FLTR) printf("%f\n", mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle]);

				ERR("_mi==_max (%f!=%f) b=%i type=%i", _min, _max, b, mdl->type_de_norme[b]);
				//ASSERT(_min != _max);
			}

			//
			FOR(0, i, N_FLTR) {
				mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i] = ( mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle] - _min)/( _max - _min );
				//if (mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i]==0) printf("%i %i %i\n", b,t,i);
			}
		};

		//#pragma omp parallel
		//#pragma omp for
		FOR(DEPART, t, FIN) {
			FOR(1, i, N_FLTR)
				mdl->dif_normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i] = mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i] - mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i-1];
			mdl->dif_normalisee[b*PRIXS*N_FLTR+t*N_FLTR+N_FLTR+0] = 0.f;
		}
	}

	mdl->normalisee__d     = cpu_vers_gpu<float>(mdl->normalisee,     BLOQUES * PRIXS * N_FLTR);
	mdl->dif_normalisee__d = cpu_vers_gpu<float>(mdl->dif_normalisee, BLOQUES * PRIXS * N_FLTR);
};

static uint * tout_zeroiser = UNIFORME_C(1);

Mdl_t * cree_mdl(
	uint T,
	uint Y[C], uint insts[C],
	ema_int_t * bloque[BLOQUES]
) {
	ASSERT(Y[C-1] == 1);
	ASSERT(Y[ 0 ] == BLOQUES * F_PAR_BLOQUES);
	ASSERT(insts[C-1] != FILTRES_PRIXS);			//	Afin d'assurer un Y=inst_VARS
	
	Mdl_t * mdl = alloc<Mdl_t>(1);

	mdl->T = T;

	//
	FOR(0, i, BLOQUES) {
		mdl->bloque[i]  = bloque[i];
		mdl->intervalles[i] = bloque[i]->intervalle;
		//
		mdl->type_de_norme[i] = bloque[i]->type_de_norme;
		mdl->min_theorique[i] = bloque[i]->min_theorique;
		mdl->max_theorique[i] = bloque[i]->max_theorique;
	};

	mdl->intervalles__d = cpu_vers_gpu<uint>(mdl->intervalles, BLOQUES);

	//
	calculer_normalisee__et__dif_normalisee(mdl);
	//raise(SIGINT);

	//	Architecture
	memcpy(mdl->insts,                 insts, sizeof(uint) * C);
	memcpy(mdl->Y,                         Y, sizeof(uint) * C);

	//	Allocation
	mdl->total_POIDS = 0;
	FOR(0, c, C) {
		if (c>0) ASSERT(insts[c] != 0);
		if (c==0) ASSERT(Y[c]==BLOQUES*F_PAR_BLOQUES);
		ASSERT(Y[c] <= MAX_Y);
		//
		cree_inst[insts[c]](mdl, c);
		//
		//mdl->p [c] = alloc<float>(mdl->inst_POIDS[c]);
		//
		mdl->p__d [c] = cpu_vers_gpu<float>(mdl->p[c], mdl->inst_POIDS[c]);
		mdl->y__d [c] = cudalloc<float>(mdl->inst_VARS [c] * T * MEGA_T);
		mdl->l__d [c] = cudalloc<float>(mdl->inst_LOCDS[c] * T * MEGA_T);
		mdl->dy__d[c] = cudalloc<float>(mdl->inst_VARS [c] * T * MEGA_T);
		mdl->dp__d[c] = cudalloc<float>(mdl->inst_POIDS[c]);
		//
		mdl->total_POIDS += mdl->inst_POIDS[c];

		if (c == C-1) {
			mdl->u_max = cudalloc<float>(T);
			mdl->u_min = cudalloc<float>(T);
		}
	}
	ASSERT(mdl->inst_DEPART_SORTIE[C-1] == 0);
	//
	mdl_normer_les_filtres(mdl);
	//
	mdl_zero_deriv_gpu(mdl, tout_zeroiser);
	//
	return mdl;
};

PAS_OPTIMISER()
void mdl_verif(Mdl_t * mdl) {
	uint nombre;

	//	---
	nombre=0;
	FOR(0, i, BLOQUES * PRIXS * N_FLTR) if (isnan(mdl->normalisee[i])) nombre++;
	printf("mdl->normalisee a     \033[93m%i\033[0m / %i nan\n", nombre, BLOQUES * PRIXS * N_FLTR);

	//	---
	nombre=0;
	FOR(0, i, BLOQUES * PRIXS * N_FLTR) if (isnan(mdl->dif_normalisee[i])) nombre++;
	printf("mdl->dif_normalisee a \033[93m%i\033[0m / %i nan\n", nombre, BLOQUES * PRIXS * N_FLTR);

	FOR(0, c, C) {
		printf("c = %i :\n", c);

		//	---
		nombre=0;
		FOR(0, i, mdl->inst_POIDS[c]) if (isnan(mdl->p[c][i])) nombre++;
		printf("  mdl->p a     \033[93m%i\033[0m / %i nan\n", nombre, mdl->inst_POIDS[c]);

		//	---
		float * y = gpu_vers_cpu<float>(mdl->y__d[c], mdl->inst_VARS[c]);
		nombre=0;
		FOR(0, i, mdl->inst_VARS[c]) if (isnan(y[i])) nombre++;
		printf("  mdl->y__d a  \033[93m%i\033[0m / %i nan\n", nombre, mdl->inst_VARS[c]);
		free(y);

		//	---
		float * l = gpu_vers_cpu<float>(mdl->l__d[c], mdl->inst_LOCDS[c]);
		nombre=0;
		FOR(0, i, mdl->inst_LOCDS[c]) if (isnan(l[i])) nombre++;
		printf("  mdl->l__d a  \033[93m%i\033[0m / %i nan\n", nombre, mdl->inst_LOCDS[c]);
		free(l);

		//	---
		float * dy = gpu_vers_cpu<float>(mdl->dy__d[c], mdl->inst_VARS[c]);
		nombre=0;
		FOR(0, i, mdl->inst_VARS[c]) if (isnan(dy[i])) nombre++;
		printf("  mdl->dy__d a \033[93m%i\033[0m / %i nan\n", nombre, mdl->inst_VARS[c]);
		free(dy);

		//	---
		float * dp = gpu_vers_cpu<float>(mdl->dp__d[c], mdl->inst_POIDS[c]);
		nombre=0;
		FOR(0, i, mdl->inst_POIDS[c]) if (isnan(dp[i])) nombre++;
		printf("  mdl->dp__d a \033[93m%i\033[0m / %i nan\n", nombre, mdl->inst_POIDS[c]);
		free(dp);
	}
};

PAS_OPTIMISER()
void mdl_re_cree_poids(Mdl_t * mdl) {
	//	Allocation
	mdl->total_POIDS = 0;
	FOR(0, c, C) {
		if (c>0) ASSERT(mdl->insts[c] != 0);
		ASSERT(mdl->Y[c] <= MAX_Y);
		//
		free(mdl->p[c]);
		CONTROLE_CUDA(hipFree(mdl->p__d[c]));
		//
		cree_inst[mdl->insts[c]](mdl, c);
		//
		//mdl->p [c] = alloc<float>(mdl->inst_POIDS[c]);
		//
		mdl->p__d [c] = cpu_vers_gpu<float>(mdl->p[c], mdl->inst_POIDS[c]);
		//
		mdl->total_POIDS += mdl->inst_POIDS[c];
	}
};

PAS_OPTIMISER()
void mdl_changer_couche_Y(Mdl_t * mdl, uint c, uint nouveau_Y) {
	mdl->total_POIDS -= mdl->inst_POIDS[c];
	{
		if (c>0) ASSERT(mdl->insts[c] != 0);
		mdl->Y[c] = nouveau_Y;
		ASSERT(mdl->Y[c] <= MAX_Y);
		//
		free(mdl->p[c]);
		CONTROLE_CUDA(hipFree(mdl->p__d[c]));
		//
		cree_inst[mdl->insts[c]](mdl, c);
		//
		//mdl->p [c] = alloc<float>(mdl->inst_POIDS[c]);
		//
		mdl->p__d [c] = cpu_vers_gpu<float>(mdl->p[c], mdl->inst_POIDS[c]);
		//
		mdl->total_POIDS += mdl->inst_POIDS[c];
	}
};

void mdl_normer_les_filtres(Mdl_t * mdl) {
	FOR(0, b, BLOQUES) {
		FOR(0, f, F_PAR_BLOQUES) {
			float min, max;
			uint type_norme = mdl->type_de_norme[b];
			if (type_norme == NORME_CLASSIQUE) {
				max=mdl->p[0][b*F_PAR_BLOQUES*N + f*N+0];
				min=mdl->p[0][b*F_PAR_BLOQUES*N + f*N+0];
				FOR(1, i, N) {
					if (max < mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i]) max = mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i];
					if (min > mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i]) min = mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i];
				}
			} else if (type_norme == NORME_THEORIQUE || type_norme == NORME_RELATIVE) {
				min = 0.0;	//se sont des filtres
				max = 1.0;	//pas les natures

				//	Borne au cas ou
				FOR(0, i, N) {
					float f_val = mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i];
					mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i] = MAX2(MIN2(f_val, max), min);
				}
			} else {
				ERR("mdl->type_de_norme[b]=%i", mdl->type_de_norme[b]);
			}
			//
			FOR(0, i, N) mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i] = (mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i]-min)/(max-min);
		};
	}
	CONTROLE_CUDA(hipMemcpy(mdl->p__d[0], mdl->p[0], sizeof(float)*BLOQUES*F_PAR_BLOQUES*N, hipMemcpyHostToDevice))
};

PAS_OPTIMISER()
void mdl_poids_cpu_vers_gpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p__d[c],  mdl->p[c],  sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyHostToDevice));
	}
};

PAS_OPTIMISER()
void mdl_poids_gpu_vers_cpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p[c],  mdl->p__d[c],  sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyDeviceToHost));
	}
};

PAS_OPTIMISER()
void mdl_gpu_vers_cpu(Mdl_t * mdl) {
	mdl_poids_gpu_vers_cpu(mdl);
}

PAS_OPTIMISER()
void mdl_cpu_vers_gpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p__d[c],  mdl->p[c],  sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyHostToDevice));
	}
};

PAS_OPTIMISER()
void liberer_mdl(Mdl_t * mdl) {
	FOR(0, c, C) {
		free(mdl->p [c]);
		//
		CONTROLE_CUDA(hipFree(mdl->p__d [c]));
		CONTROLE_CUDA(hipFree(mdl->y__d [c]));
		CONTROLE_CUDA(hipFree(mdl->l__d [c]));
		CONTROLE_CUDA(hipFree(mdl->dy__d[c]));
		CONTROLE_CUDA(hipFree(mdl->dp__d[c]));
		//
		//FOR(0, i, mdl->inst_p_separateurs[c]) free(mdl->char_inst_p_separateurs[c][i]);
		//free(mdl->mdl->char_inst_p_separateurs[c]);
	}
};

PAS_OPTIMISER()
void mdl_zero_gpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemset(mdl->y__d [c], 0, sizeof(float) * mdl->inst_VARS [c] * mdl->T * MEGA_T));
	}
};

PAS_OPTIMISER()
void mdl_zero_deriv_gpu(Mdl_t * mdl, uint zeroiser[C]) {
	FOR(0, c, C) {
		if (zeroiser[c]) {
			CONTROLE_CUDA(hipMemset(mdl->dy__d[c], 0, sizeof(float) * mdl->inst_VARS [c] * mdl->T * MEGA_T));
			CONTROLE_CUDA(hipMemset(mdl->dp__d[c], 0, sizeof(float) * mdl->inst_POIDS[c]));
		}
	}
};