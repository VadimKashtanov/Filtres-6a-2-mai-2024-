#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

float mdl_les_gains(Mdl_t * mdl, uint t0, uint t1, uint mode, float GRAND_COEF, uint _t_MODE, uint GRAINE) {
	ASSERT(GRAND_COEF >= 2);
	//
	float * _y = gpu_vers_cpu<float>(mdl->y__d[C-1], mdl->T*MEGA_T*1);
	//
	float somme     = 0;
	float potentiel = 0;
	//
	FOR(0, t, mdl->T) {
		FOR(0, mega_t, MEGA_T) {
			uint depart_plus_t = t_MODE(
				_t_MODE, GRAINE,
				t0, t0+mdl->T*MEGA_T,
				t, mega_t,
				mdl->T, MEGA_T
			);
			//
			float p1p0 = (MARCHEE_DE_TRADE[depart_plus_t+1]/MARCHEE_DE_TRADE[depart_plus_t]-1);
			//
			uint a_t_il_predit = (signe(p1p0) == signe(_y[(mega_t*mdl->T + t)*1 + 0]));
			//
			somme     += powf(fabs(p1p0),GRAND_COEF) * a_t_il_predit;
			potentiel += powf(fabs(p1p0),GRAND_COEF) * true         ;
		}
	}
	free(_y);
	return somme / potentiel;
};

float mdl_score(Mdl_t * mdl, uint t0, uint t1, uint mode, uint _t_MODE, uint GRAINE) {
	uint EXACTE = 1;
	mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode, _t_MODE, GRAINE);
	//
	float somme_score = nvidia_somme_score(EXACTE, mdl->y__d[C-1], mdl->u_max, mdl->u_min, t0, mdl->T, _t_MODE, GRAINE);
	//
	return nvidia_score_finale(somme_score, mdl->T, _t_MODE, GRAINE);
};

float mdl_pred(Mdl_t * mdl, uint t0, uint t1, uint mode, uint _t_MODE, uint GRAINE) {
	mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode, _t_MODE, GRAINE);
	return nvidia_prediction(mdl->y__d[C-1], t0, mdl->T, _t_MODE, GRAINE);
};

void mdl_aller_retour(Mdl_t * mdl, uint t0, uint t1, uint mode, uint _t_MODE, uint GRAINE, uint EXACTE) {
	mdl_zero_gpu(mdl);
	mdl_f(mdl, t0, t1, mode, _t_MODE, GRAINE);
	//
	//
	float somme_score = nvidia_somme_score(EXACTE, mdl->y__d[C-1], mdl->u_max, mdl->u_min, t0, mdl->T, _t_MODE, GRAINE);
	//
	//
	float d_score = d_nvidia_score_finale(somme_score, mdl->T, _t_MODE, GRAINE);
	//
	//
	d_nvidia_somme_score(EXACTE, d_score, mdl->y__d[C-1], mdl->dy__d[C-1], mdl->u_max, mdl->u_min, t0, mdl->T, _t_MODE, GRAINE);
	mdl_df(mdl, t0, t1, mode, _t_MODE, GRAINE);
};