#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void plume_separateur_p(Mdl_t * mdl, uint c, uint p) {
	FOR(0, i, mdl->inst_p_separateurs[c]) {
		if (p == mdl->depart_inst_p_separateurs[c][i]) {
			printf(" = \033[93m%s\033[0m =\n", mdl->char_inst_p_separateurs[c][i]);
		}
	}
};

void mdl_plume_poids(Mdl_t * mdl) {
	FOR(0, c, C) {
		printf("=========== C= %i ==========\n", c);
		plume_inst[mdl->insts[c]](mdl, c);
	}
};

void plumer_mdl(Mdl_t * mdl) {
	printf("\033[4m Plume mdl\033[0m\n");
	printf("Filtres : %i\n", mdl->Y[0]);
	printf("Lignes des bloques (pour filtres_prixs): \n");
	FOR(0, i, BLOQUES) {
		printf("%3.i| %s  source=%s intervalle=%3.i K_ema=%3.i ",
			i,
			nom_natures[mdl->bloque[i]->nature],
			nom_sources[mdl->bloque[i]->source],
			mdl->intervalles[i],
			mdl->bloque[i]->K_ema
		);
		FOR(0, j, MAX_PARAMS) printf("P%i=%3.i ", j, mdl->bloque[i]->params[j]);
		printf("\n");
	}
	printf("Instructions : \n");
	uint POIDS = 0;
	FOR(0, c, C) {
		POIDS += mdl->inst_POIDS[c];
		printf("%2.i| %s:<Y=%4.i> [poids=%7.i]\n", c, nom_inst[mdl->insts[c]], mdl->Y[c], mdl->inst_POIDS[c]);
	}
	printf("Quantitée poids = %s\n", scientifique(POIDS));
	printf(" --- fin plume mdl ---\n");
};

void comportement(Mdl_t * mdl, uint t0, uint t1) {
	uint MODE_t_MODE = t_CONTINUE;
	uint grain_t_MODE = 0;
	//
	mdl_f(mdl, t0, t1, MODE_MAXIMALE, MODE_t_MODE, grain_t_MODE);
	mdl_gpu_vers_cpu(mdl);
	FOR(0, c, C) {
		float * y = gpu_vers_cpu<float>(mdl->y__d[c], mdl->inst_VARS[c]*mdl->T*MEGA_T);
		printf("###=== c=%i ===###\n", c);
		FOR(0, i, mdl->inst_VARS[c]) {
			printf("%3.i| ", i);
			//FOR(t0, t, t1)
			uint t = t0;
			FOR(0, mega_t, MEGA_T)
				printf("%+f ", y[(mega_t*mdl->T + t-t0)*mdl->inst_VARS[c] + i]);
			printf("\n");
		}
		free(y);
	}
};

static uint * zeroiser_nulle = UNIFORME_C(1);

void mdl_plume_grad(Mdl_t * mdl, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	//uint * zeroiser_nulle = UNIFORME_C(1);
	//
	mdl_zero_deriv_gpu(mdl, zeroiser_nulle);
	//
	uint EXACTE = 1;
	mdl_aller_retour(mdl, t0, t1, MODE_MAXIMALE, _t_MODE, GRAINE, EXACTE);
	//
	FOR(0, c, C) {
		printf(" Couche %2.i \033[93mY=%5.i poids=%7.i\033[0m", c, mdl->Y[c], mdl->inst_POIDS[c]);
		uint POIDS = mdl->inst_POIDS[c];
		float * _grad = gpu_vers_cpu<float>(mdl->dp__d[c], POIDS);
		float * ____p = gpu_vers_cpu<float>(mdl->p__d[c], POIDS);
		//
		//	Grad
		//
		float moyenne = 0;
		float min=_grad[0], max=_grad[0];
		float abs_min=fabs(____p[0]);
		FOR(0, i, POIDS) {
			if (_grad[i] > max) max = _grad[i];
			if (_grad[i] < min) min = _grad[i];
			moyenne += fabs(_grad[i]);
			if (fabs(_grad[i]) < abs_min) abs_min = fabs(_grad[i]);
		}
		printf("[Grad:Min=%+f;Max=%+f;Moyenne=(-/+)%f]",
			min, max, moyenne / POIDS
		);
		free(_grad);
		//
		//	P
		//
		moyenne = 0;
		min=____p[0], max=____p[0];
		abs_min=fabs(____p[0]);
		FOR(0, i, POIDS) {
			if (____p[i] > max) max = ____p[i];
			if (____p[i] < min) min = ____p[i];
			moyenne += fabs(____p[i]);
			if (fabs(____p[i]) < abs_min) abs_min = fabs(____p[i]);
		}
		printf("[P:Min=%+f;Max=%+f;Moyenne=(-/+)%f]",
			min, max, moyenne / POIDS
		);
		free(____p);
		printf("  %s\n", nom_inst[mdl->insts[c]]);
	}
};