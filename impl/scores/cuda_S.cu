#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

//	   (a-min)/(max-min) + (b-min)/(max-min) + (c-min)/(max-min)
//	= ( (a-min)+(b-min)+(c-min)) / (max - min)
//	= ( a+b+c - 3*min) / (max-min)

#define L 3

static __global__ void kerd_nvidia_score_somme(
	uint EXACTE,
	uint _t_MODE, uint GRAINE,
	float * y,
	float * _s_max, float * _s_min,
	uint t0, uint T,
	float * score, float * _PRIXS)
{
	uint t = threadIdx.x + blockIdx.x * blockDim.x;
	//
	if (t < T) {
		float s = 0;	//******:!!!!!! normer u_max, u_min. mettre =1 quand y=0
		//				//  puis resoudre les nan
		float u = 1.0;
		float s_max;
		float s_min;
		//
		FOR(0, mega_t, MEGA_T) {
			uint depart_plus_t = t_MODE(
				_t_MODE, GRAINE,
				t0, t0+T*MEGA_T,
				t, mega_t,
				T, MEGA_T
			);

			float p1 = _PRIXS[depart_plus_t+1];
			float p0 = _PRIXS[depart_plus_t  ];

			float _y = y[(mega_t*T + t)*1 + 0];
			//
			uint _alea = PSEUDO_ALEA_cuda((1+GRAINE+mega_t*T+t));
			float alea = 2*((float)(_alea%1000))/1000.0 - 1.0;
			if (EXACTE) alea = 0;
			//
			u += u * _y * (p1/p0-1) * L;
			if (u < 0) u = 0.0;
			//
			//
			float _s = cuda_SCORE(_y, p1, p0, alea, u);
			s += _s;
			//
			if (mega_t == 0) {
				s_max = _s;
				s_min = _s;
			}
			//
			if (s_max < _s) s_max = _s;
			if (s_min > _s) s_min = _s;
		}
		//
		//if (s_max == s_min) s_min = 0;
		//
		_s_max[t] = s_max;
		_s_min[t] = s_min;
		//
		score[t] = s;///s_max;///s_max;//(s - MEGA_T*s_min) / (s_max - s_min);
		//printf("[%i,%f],\n", t, 1/s_max);
	}
};

#define HORIZON 32

static __global__ void kerd_addition_horizontale(
	float * vecteur, uint T, float * somme_finale)
{
	uint thx = threadIdx.x;
	uint t = threadIdx.x + blockIdx.x * blockDim.x;
	//
	uint __BLOQUE = blockDim.x;
	//
	if (t < T) {
		uint depart_bloque = 2*(t - (t% __BLOQUE));
		//
		for (uint mul=1; mul <= HORIZON;) {
			if (thx % mul == 0) {
				// a = b + c
				uint a = depart_bloque + 2*thx;
				uint b = depart_bloque + 2*thx;
				uint c = depart_bloque + 2*thx + 2*(mul)/2;
				//
				if (!(a < T)) assert(0);
				if (!(b < T)) assert(0);
				if (!(c < T)) assert(0);
				//
				vecteur[a] = vecteur[b] + vecteur[c];
			}
			__syncthreads();
			mul *= 2;
		}
		//
		if (thx == 0) atomicAdd(&somme_finale[0], vecteur[depart_bloque+0]);
	};
};

float nvidia_somme_score(
	uint EXACTE,
	float * y,
	float * s_max, float * s_min,
	uint depart, uint T,
	uint _t_MODE, uint GRAINE)
{
	ASSERT(T % (HORIZON*2) == 0);
	//
	float * somme_score__d = cudalloc<float>(T);
	float * somme_score_finale__d = cudalloc<float>(1);
	CONTROLE_CUDA(hipMemset(somme_score_finale__d, 0, sizeof(float)*1));
	CONTROLE_CUDA(hipMemset(somme_score__d, 0, sizeof(float)*T));
	
	//	--- Calcule du Score ---
	kerd_nvidia_score_somme<<<dim3(KERD(T,1)),dim3(1)>>>(
		EXACTE,
		_t_MODE, GRAINE,
		y,
		s_max, s_min,
		depart, T,
		somme_score__d, cuda_MARCHEE_DE_TRADE
	);
	ATTENDRE_CUDA();

	//	--- Somme Horizontale ---
	kerd_addition_horizontale<<<dim3(KERD(T/2,HORIZON)),dim3(HORIZON)>>>(
		somme_score__d,
		T, somme_score_finale__d
	);
	ATTENDRE_CUDA();

	//	Gpu vers Cpu
	float * somme_score = gpu_vers_cpu<float>(somme_score_finale__d, 1);
	float somme = somme_score[0];
	//
	CONTROLE_CUDA(hipFree(somme_score__d));
	CONTROLE_CUDA(hipFree(somme_score_finale__d));
	free(somme_score);
	//
	return somme;
};

float  nvidia_score_finale(float somme, uint T, uint _t_MODE, uint GRAINE) {
	float moyenne = somme / (float)(1 * T * MEGA_T);
	return APRES_SCORE(moyenne);
};

//	===============================================================

float d_nvidia_score_finale(float somme, uint T, uint _t_MODE, uint GRAINE) {
	float moyenne = somme / (float)(1 * T * MEGA_T);
	return dAPRES_SCORE(moyenne) / (float)(1 * T * MEGA_T);
};

//	===============================================================

static __global__ void kerd_nvidia_score_dpowf(
	uint EXACTE,
	uint _t_MODE, uint GRAINE,
	float dS, float * y, float * dy,
	float * _s_max, float * _s_min,
	uint t0, uint T,
	float * _PRIXS)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;

	if (_t < T) {
		float u = 1.0;
		//
		float s_max = _s_max[_t];
		float s_min = _s_min[_t];
		//
		//printf("(%i,%f),\n", _t, 1/s_max);
		//
		FOR(0, mega_t, MEGA_T) {
			uint depart_plus_t = t_MODE(
				_t_MODE, GRAINE,
				t0, t0+T*MEGA_T,
				_t, mega_t,
				T, MEGA_T
			);
			//
			float p1 = _PRIXS[depart_plus_t+1];
			float p0 = _PRIXS[depart_plus_t  ];
			//
			float _y = y[(mega_t*T + _t)*1 + 0];
			//
			uint _alea = PSEUDO_ALEA_cuda((1+GRAINE+mega_t*T+_t));
			float alea = 2*((float)(_alea%1000))/1000.0 - 1.0;
			if (EXACTE) alea = 0;
			//
			u += u * _y * (p1/p0 - 1) * L;
			if (u < 0) u = 0.0;
			//
			//printf("t=%i %f\n", _t, 1/s_max);
			float _dy = cuda_dSCORE(_y, p1, p0, alea, u);///s_max;// / (s_max - s_min);
			//
			atomicAdd(&dy[(mega_t*T + _t)*1+0], dS * _dy);
			//atomicAdd car certaines fonction prennent y[-1] comme entree
		}
	}
};

void d_nvidia_somme_score(
	uint EXACTE,
	float d_score,
	float * y, float * dy,
	float * u_max, float * u_min,
	uint depart, uint T,
	uint _t_MODE, uint GRAINE)
{
	kerd_nvidia_score_dpowf<<<dim3(KERD(T,1)), dim3(1)>>>(
		EXACTE,
		_t_MODE, GRAINE,
		d_score,
		y, dy,
		u_max, u_min,
		depart, T,
		cuda_MARCHEE_DE_TRADE
	);
	ATTENDRE_CUDA();
};