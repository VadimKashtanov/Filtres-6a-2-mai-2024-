#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

static __global__ void kerd_nvidia_prediction_somme(
	uint _t_MODE, uint GRAINE,
	float * y, uint t0, uint T,
	float * pred, float * _PRIXS)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	if (thx < T) {
		FOR(0, mega_t, MEGA_T) {
			uint depart_plus_t = t_MODE(
				_t_MODE, GRAINE,
				t0, t0+T*MEGA_T,
				thx, mega_t,
				T, MEGA_T
			);
			//
			float p1 = _PRIXS[depart_plus_t+1];
			float p0 = _PRIXS[depart_plus_t];
			atomicAdd(
				pred,
				1.0*(uint)(cuda_signe((y[(mega_t*T*1 + 0+thx)*1+0])) == cuda_signe((p1/p0-1)))
			);
		}
	};
};

float nvidia_prediction(float * y, uint depart, uint T, uint _t_MODE, uint GRAINE) {
	float * pred__d = cudalloc<float>(1);
	CONTROLE_CUDA(hipMemset(pred__d, 0, 1*sizeof(float)));
	kerd_nvidia_prediction_somme<<<dim3(KERD(T,1024)),dim3(1024)>>>(
		_t_MODE, GRAINE,
		y, depart, T,
		pred__d, cuda_MARCHEE_DE_TRADE
	);
	ATTENDRE_CUDA();
	float _pred;
	CONTROLE_CUDA(hipMemcpy(&_pred, pred__d, sizeof(float)*1, hipMemcpyDeviceToHost));
	cudafree<float>(pred__d);
	return _pred / (float)(T*MEGA_T);
};